#include "hip/hip_runtime.h"
#include <iostream>

////for debug purposes
//#define PRINT_RESIDUALS_SPARSE
//#define PRINT_RESIDUALS_DENSE

#define ENABLE_EARLY_OUT

#include "GlobalDefines.h"
#include "SolverBundlingParameters.h"
#include "SolverBundlingState.h"
#include "SolverBundlingUtil.h"
#include "SolverBundlingEquations.h"
#include "SolverBundlingEquationsLie.h"
#include "SolverBundlingDenseUtil.h"
#include "../SiftGPU/CUDATimer.h"

//#include <conio.h>

#define THREADS_PER_BLOCK_DENSE_DEPTH 128
#define THREADS_PER_BLOCK_DENSE_DEPTH_FLIP 64

#define THREADS_PER_BLOCK_DENSE_OVERLAP 512


/////////////////////////////////////////////////////////////////////////
// Dense Depth Term
/////////////////////////////////////////////////////////////////////////
template<bool usePairwise>
__global__ void FindImageImageCorr_Kernel(SolverInput input, SolverState state, SolverParameters parameters)
{
	// image indices
	unsigned int i, j; // project from j to i
	if (usePairwise) {
		i = blockIdx.x; j = blockIdx.y; // all pairwise
		if (i >= j) return;
	}
	else {
		i = blockIdx.x; j = i + 1; // frame-to-frame
	}
	if (input.d_validImages[i] == 0 || input.d_validImages[j] == 0) return;

	const unsigned int tidx = threadIdx.x;
	const unsigned int subWidth = input.denseDepthWidth / parameters.denseOverlapCheckSubsampleFactor;
	const unsigned int x = (tidx % subWidth) * parameters.denseOverlapCheckSubsampleFactor;
	const unsigned int y = (tidx / subWidth) * parameters.denseOverlapCheckSubsampleFactor;
	const unsigned int idx = y * input.denseDepthWidth + x;

	if (idx < (input.denseDepthWidth * input.denseDepthHeight)) {
#ifdef USE_LIE_SPACE
		float4x4 transform = state.d_xTransformInverses[i] * state.d_xTransforms[j];
#else
		float4x4 transform_i = evalRtMat(state.d_xRot[i], state.d_xTrans[i]);
		float4x4 transform_j = evalRtMat(state.d_xRot[j], state.d_xTrans[j]);
		float4x4 invTransform_i = transform_i.getInverse();						//TODO PRECOMPUTE?
		float4x4 transform = invTransform_i * transform_j;
#endif
		//if (!computeAngleDiff(transform, 1.0f)) return; //~60 degrees //TODO HERE ANGIE
		//if (!computeAngleDiff(transform, 0.8f)) return; //~45 degrees
		if (!computeAngleDiff(transform, 0.52f)) return; //~30 degrees

		// find correspondence
		__shared__ int foundCorr[1]; foundCorr[0] = 0;
		__syncthreads();
		if (findDenseCorr(idx, input.denseDepthWidth, input.denseDepthHeight,
			parameters.denseDistThresh, transform, input.intrinsics,
			input.d_cacheFrames[i].d_depthDownsampled, input.d_cacheFrames[j].d_depthDownsampled,
			parameters.denseDepthMin, parameters.denseDepthMax)) { //i tgt, j src		//TODO PARAMS
			atomicAdd(foundCorr, 1);
		} // found correspondence
		__syncthreads();
		if (tidx == 0) {
			if (foundCorr[0] > 10) { //TODO PARAMS
				int addr = atomicAdd(state.d_numDenseOverlappingImages, 1);
				state.d_denseOverlappingImages[addr] = make_uint2(i, j);
			}
		}
	} // valid image pixel
}

__global__ void FlipJtJ_Kernel(unsigned int total, unsigned int dim, float* d_JtJ)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total) {
		const unsigned int x = idx % dim;
		const unsigned int y = idx / dim;
		if (x > y) {
			d_JtJ[y * dim + x] = d_JtJ[x * dim + y];
		}
	}
}
__global__ void FindDenseCorrespondences_Kernel(SolverInput input, SolverState state, SolverParameters parameters)
{
	const int imPairIdx = blockIdx.x; //should not go out of bounds, no need to check
	uint2 imageIndices = state.d_denseOverlappingImages[imPairIdx];
	unsigned int i = imageIndices.x;	unsigned int j = imageIndices.y;

	const unsigned int tidx = threadIdx.x;
	const unsigned int gidx = tidx * gridDim.y + blockIdx.y;

	if (gidx < (input.denseDepthWidth * input.denseDepthHeight)) {
#ifdef USE_LIE_SPACE
		float4x4 transform = state.d_xTransformInverses[i] * state.d_xTransforms[j]; //invTransform_i * transform_j
#else
		float4x4 transform_i = evalRtMat(state.d_xRot[i], state.d_xTrans[i]);
		float4x4 transform_j = evalRtMat(state.d_xRot[j], state.d_xTrans[j]);
		float4x4 invTransform_i = transform_i.getInverse();						
		float4x4 transform = invTransform_i * transform_j;
#endif
		// find correspondence
		const int numWarps = THREADS_PER_BLOCK_DENSE_DEPTH / WARP_SIZE;
		__shared__ int s_count[numWarps];
		s_count[0] = 0;
		int count = 0.0f;
		//TODO HERE ANGIE
#ifdef CUDACACHE_UCHAR_NORMALS
		if (findDenseCorr(gidx, input.denseDepthWidth, input.denseDepthHeight,
			parameters.denseDistThresh, parameters.denseNormalThresh, transform, input.intrinsics,
			input.d_cacheFrames[i].d_depthDownsampled, input.d_cacheFrames[i].d_normalsDownsampledUCHAR4,
			input.d_cacheFrames[j].d_depthDownsampled, input.d_cacheFrames[j].d_normalsDownsampledUCHAR4,
			parameters.denseDepthMin, parameters.denseDepthMax)) { //i tgt, j src
#elif defined(CUDACACHE_FLOAT_NORMALS)
		if (findDenseCorr(gidx, input.denseDepthWidth, input.denseDepthHeight,
			parameters.denseDistThresh, parameters.denseNormalThresh, transform, input.intrinsics,
			input.d_cacheFrames[i].d_depthDownsampled, input.d_cacheFrames[i].d_normalsDownsampled,
			input.d_cacheFrames[j].d_depthDownsampled, input.d_cacheFrames[j].d_normalsDownsampled,
			parameters.denseDepthMin, parameters.denseDepthMax)) { //i tgt, j src
#endif
//#ifdef CUDACACHE_UCHAR_NORMALS
//		if (findDenseCorr(gidx, input.denseDepthWidth, input.denseDepthHeight,
//			parameters.denseDistThresh, parameters.denseNormalThresh, transform, input.intrinsics,
//			input.d_cacheFrames[i].d_depthDownsampled, input.d_cacheFrames[i].d_normalsDownsampledUCHAR4,
//			input.d_cacheFrames[j].d_depthDownsampled, input.d_cacheFrames[j].d_normalsDownsampledUCHAR4,
//			parameters.denseDepthMin, parameters.denseDepthMax)) { //i tgt, j src
//#else
//		if (findDenseCorr(gidx, input.denseDepthWidth, input.denseDepthHeight,
//			parameters.denseDistThresh, parameters.denseNormalThresh, transform, input.intrinsics,
//			input.d_cacheFrames[i].d_depthDownsampled, input.d_cacheFrames[i].d_normalsDownsampled,
//			input.d_cacheFrames[j].d_depthDownsampled, input.d_cacheFrames[j].d_normalsDownsampled,
//			parameters.denseDepthMin, parameters.denseDepthMax)) { //i tgt, j src
//#endif
			//atomicAdd(&state.d_denseCorrCounts[imPairIdx], 1.0f);
			count++;
		} // found correspondence
		count = warpReduce(count);
		__syncthreads();
		if (tidx % WARP_SIZE == 0) {
			s_count[tidx / WARP_SIZE] = count;
			//atomicAdd(&state.d_denseCorrCounts[imPairIdx], count);
		}
		__syncthreads();
		for (unsigned int stride = numWarps / 2; stride > 0; stride /= 2) {
			if (tidx < stride) s_count[tidx] = s_count[tidx] + s_count[tidx + stride];
			__syncthreads();
		}
		if (tidx == 0) {
			atomicAdd(&state.d_denseCorrCounts[imPairIdx], s_count[0]);
		}
	} // valid image pixel
}

__global__ void WeightDenseCorrespondences_Kernel(unsigned int N, SolverState state)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N) {
		// apply ln to weights
		float x = state.d_denseCorrCounts[idx];
		if (x > 0) {
			//if (x < 3200) state.d_denseCorrCounts[idx] = 0; //don't consider too small #corr //TODO PARAMS
			if (x < 800) state.d_denseCorrCounts[idx] = 0; //don't consider too small #corr //TODO PARAMS
			//if (x < 400) state.d_denseCorrCounts[idx] = 0; //don't consider too small #corr //TODO PARAMS
			//if (x < 200) state.d_denseCorrCounts[idx] = 0; //don't consider too small #corr //TODO PARAMS //TODO EVAL DEBUG
			else {
				state.d_denseCorrCounts[idx] = 1.0f / min(logf(x), 9.0f); // natural log //TODO PARAMS
			}

			//state.d_denseCorrCounts[idx] = 1.0f / clamp(logf(x), 2.0f, 9.0f); // natural log //TODO PARAMS
		}
	}
}

template<bool useDepth, bool useColor>
__global__ void BuildDenseSystem_Kernel(SolverInput input, SolverState state, SolverParameters parameters)
{
	const int imPairIdx = blockIdx.x;
	uint2 imageIndices = state.d_denseOverlappingImages[imPairIdx];
	unsigned int i = imageIndices.x;	unsigned int j = imageIndices.y;

	float imPairWeight = state.d_denseCorrCounts[imPairIdx];
	if (imPairWeight == 0.0f) return;

	const unsigned int idx = threadIdx.x;
	const unsigned int srcIdx = idx * gridDim.y + blockIdx.y;

	if (srcIdx < (input.denseDepthWidth * input.denseDepthHeight)) {
#ifdef USE_LIE_SPACE
		float4x4 transform_i = state.d_xTransforms[i];
		float4x4 transform_j = state.d_xTransforms[j];
		float4x4 invTransform_i = state.d_xTransformInverses[i];
		float4x4 invTransform_j = state.d_xTransformInverses[j];
		float4x4 transform = invTransform_i * transform_j;
#else
		float4x4 transform_i = evalRtMat(state.d_xRot[i], state.d_xTrans[i]);
		float4x4 transform_j = evalRtMat(state.d_xRot[j], state.d_xTrans[j]);
		float4x4 invTransform_i = transform_i.getInverse();						//TODO PRECOMPUTE?
		float4x4 transform = invTransform_i * transform_j;
#endif
		// point-to-plane term
		matNxM<1, 6> depthJacBlockRow_i, depthJacBlockRow_j; depthJacBlockRow_i.setZero(); depthJacBlockRow_j.setZero();
		float depthRes = 0.0f; float depthWeight = 0.0f;
		// color term
		matNxM<1, 6> colorJacBlockRow_i, colorJacBlockRow_j; colorJacBlockRow_i.setZero(); colorJacBlockRow_j.setZero();
		float colorRes = 0.0f; float colorWeight = 0.0f;

		// find correspondence
		float3 camPosSrc; float3 camPosSrcToTgt; float3 camPosTgt; float3 normalTgt; float2 tgtScreenPos;
		//TODO HERE ANGIE
#ifdef CUDACACHE_FLOAT_NORMALS
		bool foundCorr = findDenseCorr(srcIdx, input.denseDepthWidth, input.denseDepthHeight,
			parameters.denseDistThresh, parameters.denseNormalThresh, transform, input.intrinsics,
			input.d_cacheFrames[i].d_cameraposDownsampled, input.d_cacheFrames[i].d_normalsDownsampled,
			input.d_cacheFrames[j].d_cameraposDownsampled, input.d_cacheFrames[j].d_normalsDownsampled,
			parameters.denseDepthMin, parameters.denseDepthMax, camPosSrc, camPosSrcToTgt, tgtScreenPos, camPosTgt, normalTgt); //i tgt, j src
#elif defined(CUDACACHE_UCHAR_NORMALS)
		bool foundCorr = findDenseCorr(srcIdx, input.denseDepthWidth, input.denseDepthHeight,
			parameters.denseDistThresh, parameters.denseNormalThresh, transform, input.intrinsics,
			input.d_cacheFrames[i].d_cameraposDownsampled, input.d_cacheFrames[i].d_normalsDownsampledUCHAR4,
			input.d_cacheFrames[j].d_cameraposDownsampled, input.d_cacheFrames[j].d_normalsDownsampledUCHAR4,
			parameters.denseDepthMin, parameters.denseDepthMax, camPosSrc, camPosSrcToTgt, tgtScreenPos, camPosTgt, normalTgt); //i tgt, j src
#endif
//#ifdef CUDACACHE_UCHAR_NORMALS
//		bool foundCorr = findDenseCorr(srcIdx, input.denseDepthWidth, input.denseDepthHeight,
//			parameters.denseDistThresh, parameters.denseNormalThresh, transform, input.intrinsics,
//			input.d_cacheFrames[i].d_cameraposDownsampled, input.d_cacheFrames[i].d_normalsDownsampledUCHAR4,
//			input.d_cacheFrames[j].d_cameraposDownsampled, input.d_cacheFrames[j].d_normalsDownsampledUCHAR4,
//			parameters.denseDepthMin, parameters.denseDepthMax, camPosSrc, camPosSrcToTgt, tgtScreenPos, camPosTgt, normalTgt); //i tgt, j src
//#else
//		bool foundCorr = findDenseCorr(srcIdx, input.denseDepthWidth, input.denseDepthHeight,
//			parameters.denseDistThresh, parameters.denseNormalThresh, transform, input.intrinsics,
//			input.d_cacheFrames[i].d_cameraposDownsampled, input.d_cacheFrames[i].d_normalsDownsampled,
//			input.d_cacheFrames[j].d_cameraposDownsampled, input.d_cacheFrames[j].d_normalsDownsampled,
//			parameters.denseDepthMin, parameters.denseDepthMax, camPosSrc, camPosSrcToTgt, tgtScreenPos, camPosTgt, normalTgt); //i tgt, j src
//#endif
		if (useDepth) {
			if (foundCorr) {
				// point-to-plane residual
				float3 diff = camPosTgt - camPosSrcToTgt;
				depthRes = dot(diff, normalTgt);
				//depthWeight = parameters.weightDenseDepth * imPairWeight * max(0.0f, 0.5f*((1.0f - length(diff) / parameters.denseDistThresh) + (1.0f - camPosTgt.z / parameters.denseDepthMax)));
				//depthWeight = parameters.weightDenseDepth * imPairWeight * max(0.0f, (1.0f - camPosTgt.z / 2.0f)); //fr1_desk
				//depthWeight = parameters.weightDenseDepth * imPairWeight * max(0.0f, (1.0f - camPosTgt.z / 2.5f)); //fr3_office, fr2_xyz_half // livingroom1
				//depthWeight = parameters.weightDenseDepth * imPairWeight * max(0.0f, (1.0f - camPosTgt.z / 3.0f)); //fr3_nstn
				//depthWeight = parameters.weightDenseDepth * imPairWeight * max(0.0f, (1.0f - camPosTgt.z / 1.8f));
				//depthWeight = parameters.weightDenseDepth * imPairWeight * (pow(max(0.0f, 1.0f - camPosTgt.z / 2.5f), 1.8f));
				//depthWeight = parameters.weightDenseDepth * imPairWeight * (pow(max(0.0f, 1.0f - camPosTgt.z / 2.0f), 1.8f)); //fr3_office, fr1_desk_f20
				depthWeight = parameters.weightDenseDepth * imPairWeight * (pow(max(0.0f, 1.0f - camPosTgt.z / 2.0f), 2.5f)); //fr2_xyz_half
				//depthWeight = parameters.weightDenseDepth * imPairWeight * (pow(max(0.0f, 1.0f - camPosTgt.z / 3.5f), 1.8f)); //fr3_nstn

				//depthWeight = parameters.weightDenseDepth * imPairWeight * (pow(max(0.0f, 1.0f - camPosTgt.z / parameters.denseDepthMax), 1.8f)); //TODO EVAL DEBUGGING

				//float wtgt = (pow(max(0.0f, 1.0f - camPosTgt.z / 2.5f), 1.8f));
				//float wsrc = (pow(max(0.0f, 1.0f - camPosSrc.z / 2.5f), 1.8f));
				//depthWeight = parameters.weightDenseDepth * imPairWeight * wtgt * wsrc;
#ifdef USE_LIE_SPACE
				if (i > 0) computeJacobianBlockRow_i(depthJacBlockRow_i, transform_i, invTransform_j, camPosSrc, normalTgt);
				if (j > 0) computeJacobianBlockRow_j(depthJacBlockRow_j, invTransform_i, transform_j, camPosSrc, normalTgt);
#else
				if (i > 0) computeJacobianBlockRow_i(depthJacBlockRow_i, state.d_xRot[i], state.d_xTrans[i], transform_j, camPosSrc, normalTgt);
				if (j > 0) computeJacobianBlockRow_j(depthJacBlockRow_j, state.d_xRot[j], state.d_xTrans[j], invTransform_i, camPosSrc, normalTgt);
#endif
			}
			addToLocalSystem(foundCorr, state.d_denseJtJ, state.d_denseJtr, input.numberOfImages * 6,
				depthJacBlockRow_i, depthJacBlockRow_j, i, j, depthRes, depthWeight, idx
				, state.d_sumResidual, state.d_corrCount);
			//addToLocalSystemBrute(foundCorr, state.d_denseJtJ, state.d_denseJtr, input.numberOfImages * 6,
			//	depthJacBlockRow_i, depthJacBlockRow_j, i, j, depthRes, depthWeight, idx);
		}
		if (useColor) {
			bool foundCorrColor = false;
			if (foundCorr) {
				const float2 intensityDerivTgt = bilinearInterpolationFloat2(tgtScreenPos.x, tgtScreenPos.y, input.d_cacheFrames[i].d_intensityDerivsDownsampled, input.denseDepthWidth, input.denseDepthHeight);
				const float intensityTgt = bilinearInterpolationFloat(tgtScreenPos.x, tgtScreenPos.y, input.d_cacheFrames[i].d_intensityDownsampled, input.denseDepthWidth, input.denseDepthHeight);
				colorRes = intensityTgt - input.d_cacheFrames[j].d_intensityDownsampled[srcIdx];
				foundCorrColor = (intensityDerivTgt.x != MINF && abs(colorRes) < parameters.denseColorThresh && length(intensityDerivTgt) > parameters.denseColorGradientMin);
				if (foundCorrColor) {
					const float2 focalLength = make_float2(input.intrinsics.x, input.intrinsics.y);
#ifdef USE_LIE_SPACE
					if (i > 0) computeJacobianBlockIntensityRow_i(colorJacBlockRow_i, focalLength, transform_i, invTransform_j, camPosSrc, camPosSrcToTgt, intensityDerivTgt);
					if (j > 0) computeJacobianBlockIntensityRow_j(colorJacBlockRow_j, focalLength, invTransform_i, transform_j, camPosSrc, camPosSrcToTgt, intensityDerivTgt);
#else
					if (i > 0) computeJacobianBlockIntensityRow_i(colorJacBlockRow_i, focalLength, state.d_xRot[i], state.d_xTrans[i], transform_j, camPosSrc, camPosSrcToTgt, intensityDerivTgt);
					if (j > 0) computeJacobianBlockIntensityRow_j(colorJacBlockRow_j, focalLength, state.d_xRot[j], state.d_xTrans[j], invTransform_i, camPosSrc, camPosSrcToTgt, intensityDerivTgt);
#endif
					colorWeight = parameters.weightDenseColor * imPairWeight * max(0.0f, 1.0f - abs(colorRes) / (1.15f*parameters.denseColorThresh));
					//colorWeight = parameters.weightDenseColor * imPairWeight * max(0.0f, 1.0f - abs(colorRes) / parameters.denseColorThresh) * max(0.0f, (1.0f - camPosTgt.z / 1.0f));
					//colorWeight = parameters.weightDenseColor * imPairWeight * max(0.0f, 0.5f*(1.0f - abs(colorRes) / parameters.denseColorThresh) + 0.5f*max(0.0f, (1.0f - camPosTgt.z / 1.0f)));
				}
			}
			addToLocalSystem(foundCorrColor, state.d_denseJtJ, state.d_denseJtr, input.numberOfImages * 6,
				colorJacBlockRow_i, colorJacBlockRow_j, i, j, colorRes, colorWeight, idx
				, state.d_sumResidualColor, state.d_corrCountColor);
			//addToLocalSystemBrute(foundCorrColor, state.d_denseJtJ, state.d_denseJtr, input.numberOfImages * 6,
			//	colorJacBlockRow_i, colorJacBlockRow_j, i, j, colorRes, colorWeight, idx);
		}
	} // valid image pixel
}

bool BuildDenseSystem(const SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer* timer)
{
	const unsigned int N = input.numberOfImages;
	const int sizeJtr = 6 * N;
	const int sizeJtJ = sizeJtr * sizeJtr;

#ifdef PRINT_RESIDUALS_DENSE
	cutilSafeCall(hipMemset(state.d_corrCount, 0, sizeof(int)));
	cutilSafeCall(hipMemset(state.d_sumResidual, 0, sizeof(float)));
	cutilSafeCall(hipMemset(state.d_corrCountColor, 0, sizeof(int)));
	cutilSafeCall(hipMemset(state.d_sumResidualColor, 0, sizeof(float)));
#endif

	const unsigned int maxDenseImPairs = input.numberOfImages * (input.numberOfImages - 1) / 2;
	cutilSafeCall(hipMemset(state.d_denseCorrCounts, 0, sizeof(float) * maxDenseImPairs));
	cutilSafeCall(hipMemset(state.d_denseJtJ, 0, sizeof(float) * sizeJtJ));
	cutilSafeCall(hipMemset(state.d_denseJtr, 0, sizeof(float) * sizeJtr));
	cutilSafeCall(hipMemset(state.d_numDenseOverlappingImages, 0, sizeof(int)));
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

	dim3 gridImImOverlap;
	if (parameters.useDenseDepthAllPairwise) gridImImOverlap = dim3(N, N, 1); // pairwise
	else gridImImOverlap = dim3(N - 1, 1, 1); // for frame-to-frame

	if (timer) timer->startEvent("BuildDenseDepthSystem - find image corr");
	if (parameters.useDenseDepthAllPairwise) FindImageImageCorr_Kernel<true> << < gridImImOverlap, THREADS_PER_BLOCK_DENSE_OVERLAP >> >(input, state, parameters);
	else									 FindImageImageCorr_Kernel<false> << < gridImImOverlap, THREADS_PER_BLOCK_DENSE_OVERLAP >> >(input, state, parameters);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	if (timer) timer->endEvent();

	int numOverlapImagePairs;
	cutilSafeCall(hipMemcpy(&numOverlapImagePairs, state.d_numDenseOverlappingImages, sizeof(int), hipMemcpyDeviceToHost));
	if (numOverlapImagePairs == 0) {
		printf("warning: no overlapping images for dense solve\n");
		return false;
	}
	const int reductionGlobal = (input.denseDepthWidth*input.denseDepthHeight + THREADS_PER_BLOCK_DENSE_DEPTH - 1) / THREADS_PER_BLOCK_DENSE_DEPTH;
	dim3 grid(numOverlapImagePairs, reductionGlobal);
	//if (N > 11) printf("num overlap image pairs = %d\n", numOverlapImagePairs); //debugging only

	if (timer) timer->startEvent("BuildDenseDepthSystem - compute im-im weights");

	FindDenseCorrespondences_Kernel << <grid, THREADS_PER_BLOCK_DENSE_DEPTH >> >(input, state, parameters);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	////debugging //remember the delete!
	//float* denseCorrCounts = new float[numOverlapImagePairs];
	//cutilSafeCall(hipMemcpy(denseCorrCounts, state.d_denseCorrCounts, sizeof(float)*numOverlapImagePairs, hipMemcpyDeviceToHost));
	//unsigned int totalCount = 0;
	//for (unsigned int i = 0; i < numOverlapImagePairs; i++) { totalCount += (unsigned int)denseCorrCounts[i]; }
	//printf("total count = %d\n", totalCount);

	//uint2* imageIndices = new uint2[numOverlapImagePairs];
	//cutilSafeCall(hipMemcpy(imageIndices, state.d_denseOverlappingImages, sizeof(uint2)*numOverlapImagePairs, hipMemcpyDeviceToHost));
	//if (imageIndices) delete[] imageIndices;
	////debugging

	//debugging - compute some overlap stats
	//if (true || input.numberOfImages > 11) {
	//	float4x4* transforms = new float4x4[input.numberOfImages];
	//	float* denseCorrCounts = new float[numOverlapImagePairs];
	//	uint2* imageIndices = new uint2[numOverlapImagePairs];
	//	cutilSafeCall(hipMemcpy(denseCorrCounts, state.d_denseCorrCounts, sizeof(float)*numOverlapImagePairs, hipMemcpyDeviceToHost));
	//	cutilSafeCall(hipMemcpy(imageIndices, state.d_denseOverlappingImages, sizeof(uint2)*numOverlapImagePairs, hipMemcpyDeviceToHost));
	//	cutilSafeCall(hipMemcpy(transforms, state.d_xTransforms, sizeof(float4x4)*input.numberOfImages, hipMemcpyDeviceToHost));
	//	FILE* fp = fopen("debug/overlaps.csv", "w");
	//	char buffer[128];
	//	for (int i = 0; i < numOverlapImagePairs; i++) {
	//		if (denseCorrCounts[i] > 0) {
	//			float3 d = transforms[imageIndices[i].x].getTranslation() - transforms[imageIndices[i].y].getTranslation();
	//			sprintf(buffer, "%d,%d,%d,%f\n", imageIndices[i].x, imageIndices[i].y, (int)denseCorrCounts[i], length(d));
	//			fwrite(buffer, sizeof(char), strlen(buffer), fp);
	//		}
	//	}
	//	fclose(fp);
	//	if (transforms) delete[] transforms;
	//	if (denseCorrCounts) delete[] denseCorrCounts;
	//	if (imageIndices) delete[] imageIndices;
	//	int a = 5;
	//}

	int wgrid = (numOverlapImagePairs + THREADS_PER_BLOCK_DENSE_DEPTH_FLIP - 1) / THREADS_PER_BLOCK_DENSE_DEPTH_FLIP;
	WeightDenseCorrespondences_Kernel << < wgrid, THREADS_PER_BLOCK_DENSE_DEPTH_FLIP >> >(maxDenseImPairs, state);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	////debugging
	//cutilSafeCall(hipMemcpy(denseCorrCounts, state.d_denseCorrCounts, sizeof(float)*maxDenseImPairs, hipMemcpyDeviceToHost));
	//totalCount = 0;
	//for (unsigned int i = 0; i < maxDenseImPairs; i++) { if (denseCorrCounts[i] > 0.0f) totalCount++; }
	//printf("total count = %d\n", totalCount);
	//if (denseCorrCounts) delete[] denseCorrCounts;
	////debugging
	if (timer) timer->endEvent();
	if (timer) timer->startEvent("BuildDenseDepthSystem - build jtj/jtr");

	if (parameters.weightDenseDepth > 0.0f) {
		if (parameters.weightDenseColor > 0.0f) BuildDenseSystem_Kernel<true, true> << <grid, THREADS_PER_BLOCK_DENSE_DEPTH >> >(input, state, parameters);
		else									BuildDenseSystem_Kernel<true, false> << <grid, THREADS_PER_BLOCK_DENSE_DEPTH >> >(input, state, parameters);
	}
	else {
		BuildDenseSystem_Kernel<false, true> << <grid, THREADS_PER_BLOCK_DENSE_DEPTH >> >(input, state, parameters);
	}
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	
	////debugging
	//bool debugPrint = true;
	//float* h_JtJ = NULL;
	//float* h_Jtr = NULL;
	//if (debugPrint) {
	//	h_JtJ = new float[sizeJtJ];
	//	h_Jtr = new float[sizeJtr];
	//	cutilSafeCall(hipMemcpy(h_JtJ, state.d_denseJtJ, sizeof(float) * sizeJtJ, hipMemcpyDeviceToHost));
	//	cutilSafeCall(hipMemcpy(h_Jtr, state.d_denseJtr, sizeof(float) * sizeJtr, hipMemcpyDeviceToHost));
	//	printf("JtJ:\n");
	//	//for (unsigned int i = 0; i < 6 * N; i++) {
	//	//	for (unsigned int j = 0; j < 6 * N; j++)
	//	for (unsigned int i = 6 * 1; i < 6 * 2; i++) {
	//		for (unsigned int j = 6 * 1; j < 6 * 2; j++)
	//			printf(" %f,", h_JtJ[j * 6 * N + i]);
	//		printf("\n");
	//	}
	//	printf("Jtr:\n");
	//	for (unsigned int i = 0; i < 6 * N; i++) {
	//		printf(" %f,", h_Jtr[i]);
	//	}
	//	printf("\n");
	//}
	////debugging
#ifdef PRINT_RESIDUALS_DENSE
	if (parameters.weightDenseDepth > 0) {
		float sumResidual; int corrCount;
		cutilSafeCall(hipMemcpy(&sumResidual, state.d_sumResidual, sizeof(float), hipMemcpyDeviceToHost));
		cutilSafeCall(hipMemcpy(&corrCount, state.d_corrCount, sizeof(int), hipMemcpyDeviceToHost));
		printf("\tdense depth: weights * residual = %f * %f = %f\t[#corr = %d]\n", parameters.weightDenseDepth, sumResidual / parameters.weightDenseDepth, sumResidual, corrCount);
	}
	if (parameters.weightDenseColor > 0) {
		float sumResidual; int corrCount;
		cutilSafeCall(hipMemcpy(&sumResidual, state.d_sumResidualColor, sizeof(float), hipMemcpyDeviceToHost));
		cutilSafeCall(hipMemcpy(&corrCount, state.d_corrCountColor, sizeof(int), hipMemcpyDeviceToHost));
		printf("\tdense color: weights * residual = %f * %f = %f\t[#corr = %d]\n", parameters.weightDenseColor, sumResidual / parameters.weightDenseColor, sumResidual, corrCount);
	}
#endif
	const unsigned int flipgrid = (sizeJtJ + THREADS_PER_BLOCK_DENSE_DEPTH_FLIP - 1) / THREADS_PER_BLOCK_DENSE_DEPTH_FLIP;
	FlipJtJ_Kernel << <flipgrid, THREADS_PER_BLOCK_DENSE_DEPTH_FLIP >> >(sizeJtJ, sizeJtr, state.d_denseJtJ);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	if (timer) timer->endEvent();
	return true;
}

//todo more efficient?? (there are multiple per image-image...)
//get high residuals
__global__ void collectHighResidualsDevice(SolverInput input, SolverState state, SolverStateAnalysis analysis, SolverParameters parameters, unsigned int maxNumHighResiduals)
{
	const unsigned int N = input.numberOfCorrespondences; // Number of block variables
	const unsigned int corrIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (corrIdx < N) {
		float residual = evalAbsMaxResidualDevice(corrIdx, input, state, parameters);
		if (residual > parameters.highResidualThresh) {
			int idx = atomicAdd(state.d_countHighResidual, 1);
			if (idx < maxNumHighResiduals) {
				analysis.d_maxResidual[idx] = residual;
				analysis.d_maxResidualIndex[idx] = corrIdx;
			}
		}
	}
}
extern "C" void collectHighResiduals(SolverInput& input, SolverState& state, SolverStateAnalysis& analysis, SolverParameters& parameters, CUDATimer* timer)
{
	if (timer) timer->startEvent(__FUNCTION__);
	cutilSafeCall(hipMemset(state.d_countHighResidual, 0, sizeof(int)));

	const unsigned int N = input.numberOfCorrespondences; // Number of correspondences 
	unsigned int maxNumHighResiduals = (input.maxCorrPerImage*input.maxNumberOfImages + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	collectHighResidualsDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, analysis, parameters, maxNumHighResiduals);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	if (timer) timer->endEvent();
}

/////////////////////////////////////////////////////////////////////////
// Eval Max Residual
/////////////////////////////////////////////////////////////////////////

__global__ void EvalMaxResidualDevice(SolverInput input, SolverState state, SolverStateAnalysis analysis, SolverParameters parameters)
{
	__shared__ int maxResIndex[THREADS_PER_BLOCK];
	__shared__ float maxRes[THREADS_PER_BLOCK];

	const unsigned int N = input.numberOfCorrespondences; // Number of block variables
	const unsigned int corrIdx = blockIdx.x * blockDim.x + threadIdx.x;

	maxResIndex[threadIdx.x] = 0;
	maxRes[threadIdx.x] = 0.0f;

	if (corrIdx < N) {
		float residual = evalAbsMaxResidualDevice(corrIdx, input, state, parameters);

		maxRes[threadIdx.x] = residual;
		maxResIndex[threadIdx.x] = corrIdx;

		__syncthreads();

		for (int stride = THREADS_PER_BLOCK / 2; stride > 0; stride /= 2) {

			if (threadIdx.x < stride) {
				int first = threadIdx.x;
				int second = threadIdx.x + stride;
				if (maxRes[first] < maxRes[second]) {
					maxRes[first] = maxRes[second];
					maxResIndex[first] = maxResIndex[second];
				}
			}

			__syncthreads();
		}

		if (threadIdx.x == 0) {
			//printf("d_maxResidual[%d] = %f (index %d)\n", blockIdx.x, maxRes[0], maxResIndex[0]);
			analysis.d_maxResidual[blockIdx.x] = maxRes[0];
			analysis.d_maxResidualIndex[blockIdx.x] = maxResIndex[0];
		}
	}
}

extern "C" void evalMaxResidual(SolverInput& input, SolverState& state, SolverStateAnalysis& analysis, SolverParameters& parameters, CUDATimer* timer)
{
	if (timer) timer->startEvent(__FUNCTION__);

	const unsigned int N = input.numberOfCorrespondences; // Number of correspondences 
	EvalMaxResidualDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, analysis, parameters);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	if (timer) timer->endEvent();
}

/////////////////////////////////////////////////////////////////////////
// Eval Cost
/////////////////////////////////////////////////////////////////////////

__global__ void ResetResidualDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x == 0) state.d_sumResidual[0] = 0.0f;
}

__global__ void EvalResidualDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfCorrespondences; // Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	float residual = 0.0f;
	if (x < N) {
		residual = evalFDevice(x, input, state, parameters);
		//float out = warpReduce(residual);
		//unsigned int laneid;
		////This command gets the lane ID within the current warp
		//asm("mov.u32 %0, %%laneid;" : "=r"(laneid));
		//if (laneid == 0) {
		//	atomicAdd(&state.d_sumResidual[0], out);
		//}
		atomicAdd(&state.d_sumResidual[0], residual);
	}
}

extern "C" float EvalResidual(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer* timer)
{
	if (timer) timer->startEvent(__FUNCTION__);

	float residual = 0.0f;

	const unsigned int N = input.numberOfCorrespondences; // Number of block variables
	ResetResidualDevice << < 1, 1, 1 >> >(input, state, parameters);
	EvalResidualDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);

	residual = state.getSumResidual();

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	if (timer) timer->endEvent();

	return residual;
}

/////////////////////////////////////////////////////////////////////////
// Eval Linear Residual
/////////////////////////////////////////////////////////////////////////

//__global__ void SumLinearResDevice(SolverInput input, SolverState state, SolverParameters parameters)
//{
//	const unsigned int N = input.numberOfImages; // Number of block variables
//	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
//
//	float residual = 0.0f;
//	if (x > 0 && x < N) {
//		residual = dot(state.d_rRot[x], state.d_rRot[x]) + dot(state.d_rTrans[x], state.d_rTrans[x]);
//		atomicAdd(state.d_sumLinResidual, residual);
//	}
//}
//float EvalLinearRes(SolverInput& input, SolverState& state, SolverParameters& parameters)
//{
//	float residual = 0.0f;
//
//	const unsigned int N = input.numberOfImages;	// Number of block variables
//
//	// Do PCG step
//	const int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
//
//	float init = 0.0f;
//	cutilSafeCall(hipMemcpy(state.d_sumLinResidual, &init, sizeof(float), hipMemcpyHostToDevice));
//
//	SumLinearResDevice << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state, parameters);
//#ifdef _DEBUG
//	cutilSafeCall(hipDeviceSynchronize());
//	cutilCheckMsg(__FUNCTION__);
//#endif
//
//	cutilSafeCall(hipMemcpy(&residual, state.d_sumLinResidual, sizeof(float), hipMemcpyDeviceToHost));
//	return residual;
//}

/////////////////////////////////////////////////////////////////////////
// Count High Residuals
/////////////////////////////////////////////////////////////////////////

__global__ void CountHighResidualsDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfCorrespondences; // Number of block variables
	const unsigned int corrIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (corrIdx < N) {
		float residual = evalAbsMaxResidualDevice(corrIdx, input, state, parameters);

		if (residual > parameters.verifyOptDistThresh)
			atomicAdd(state.d_countHighResidual, 1);
	}
}

extern "C" int countHighResiduals(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer* timer)
{
	if (timer) timer->startEvent(__FUNCTION__);

	const unsigned int N = input.numberOfCorrespondences; // Number of correspondences
	cutilSafeCall(hipMemset(state.d_countHighResidual, 0, sizeof(int)));
	CountHighResidualsDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);

	int count;
	cutilSafeCall(hipMemcpy(&count, state.d_countHighResidual, sizeof(int), hipMemcpyDeviceToHost));
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

	if (timer) timer->endEvent();
	return count;
}

/////////////////////////////////////////////////////////////////////////
// Convergence Analysis
/////////////////////////////////////////////////////////////////////////

//uses same data store as max residual
__global__ void EvalGNConvergenceDevice(SolverInput input, SolverStateAnalysis analysis, SolverState state) //compute max of delta
{
	__shared__ float maxVal[THREADS_PER_BLOCK];

	const unsigned int N = input.numberOfImages;
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	maxVal[threadIdx.x] = 0.0f;

	if (x < N)
	{
		if (x == 0 || input.d_validImages[x] == 0)
			maxVal[threadIdx.x] = 0.0f;
		else {
			float3 r3 = fmaxf(fabs(state.d_deltaRot[x]), fabs(state.d_deltaTrans[x]));
			float r = fmaxf(r3.x, fmaxf(r3.y, r3.z));
			maxVal[threadIdx.x] = r;
		}
		__syncthreads();

		for (int stride = THREADS_PER_BLOCK / 2; stride > 0; stride /= 2) {
			if (threadIdx.x < stride) {
				int first = threadIdx.x;
				int second = threadIdx.x + stride;
				maxVal[first] = fmaxf(maxVal[first], maxVal[second]);
			}
			__syncthreads();
		}
		if (threadIdx.x == 0) {
			analysis.d_maxResidual[blockIdx.x] = maxVal[0];
		}
	}
}
float EvalGNConvergence(SolverInput& input, SolverState& state, SolverStateAnalysis& analysis, CUDATimer* timer)
{
	if (timer) timer->startEvent(__FUNCTION__);

	const unsigned int N = input.numberOfImages;
	const unsigned int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	EvalGNConvergenceDevice << < blocksPerGrid, THREADS_PER_BLOCK >> >(input, analysis, state);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	//copy to host and compute max
	cutilSafeCall(hipMemcpy(analysis.h_maxResidual, analysis.d_maxResidual, sizeof(float) * blocksPerGrid, hipMemcpyDeviceToHost));
	cutilSafeCall(hipMemcpy(analysis.h_maxResidualIndex, analysis.d_maxResidualIndex, sizeof(int) * blocksPerGrid, hipMemcpyDeviceToHost));
	float maxVal = 0.0f;
	for (unsigned int i = 0; i < blocksPerGrid; i++) {
		if (maxVal < analysis.h_maxResidual[i]) maxVal = analysis.h_maxResidual[i];
	}
	if (timer) timer->endEvent();

	return maxVal;
}

// For the naming scheme of the variables see:
// http://en.wikipedia.org/wiki/Conjugate_gradient_method
// This code is an implementation of their PCG pseudo code

template<bool useDense>
__global__ void PCGInit_Kernel1(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfImages;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	float d = 0.0f;
	if (x > 0 && x < N)
	{
		float3 resRot, resTrans;
		evalMinusJTFDevice<useDense>(x, input, state, parameters, resRot, resTrans);  // residuum = J^T x -F - A x delta_0  => J^T x -F, since A x x_0 == 0 

		state.d_rRot[x] = resRot;											// store for next iteration
		state.d_rTrans[x] = resTrans;										// store for next iteration

		const float3 pRot = state.d_precondionerRot[x] * resRot;			// apply preconditioner M^-1
		state.d_pRot[x] = pRot;

		const float3 pTrans = state.d_precondionerTrans[x] * resTrans;		// apply preconditioner M^-1
		state.d_pTrans[x] = pTrans;

		d = dot(resRot, pRot) + dot(resTrans, pTrans);						// x-th term of nomimator for computing alpha and denominator for computing beta

		state.d_Ap_XRot[x] = make_float3(0.0f, 0.0f, 0.0f);
		state.d_Ap_XTrans[x] = make_float3(0.0f, 0.0f, 0.0f);
	}

	d = warpReduce(d);
	if (threadIdx.x % WARP_SIZE == 0)
	{
		atomicAdd(state.d_scanAlpha, d);
	}
}

__global__ void PCGInit_Kernel2(unsigned int N, SolverState state)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x > 0 && x < N) state.d_rDotzOld[x] = state.d_scanAlpha[0];				// store result for next kernel call
}

void Initialization(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer* timer)
{
	const unsigned int N = input.numberOfImages;

	const int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	if (blocksPerGrid > THREADS_PER_BLOCK)
	{
		std::cout << "Too many variables for this block size. Maximum number of variables for two kernel scan: " << THREADS_PER_BLOCK*THREADS_PER_BLOCK << std::endl;
		while (1);
	}

	if (timer) timer->startEvent("Initialization");

	//!!!DEBUGGING //remember to uncomment the delete...
	//float3* rRot = new float3[input.numberOfImages]; // -jtf
	//float3* rTrans = new float3[input.numberOfImages];
	//!!!DEBUGGING

	cutilSafeCall(hipMemset(state.d_scanAlpha, 0, sizeof(float)));
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif		

	if (parameters.useDense) PCGInit_Kernel1<true> << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state, parameters);
	else PCGInit_Kernel1<false> << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state, parameters);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif		

	//cutilSafeCall(hipMemcpy(rRot, state.d_rRot, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));
	//cutilSafeCall(hipMemcpy(rTrans, state.d_rTrans, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));
	//for (unsigned int i = 1; i < input.numberOfImages; i++) { if (isnan(rRot[i].x)) { printf("NaN in jtr rRot %d\n", i); getchar(); } }
	//for (unsigned int i = 1; i < input.numberOfImages; i++) { if (isnan(rTrans[i].x)) { printf("NaN in jtr rTrans %d\n", i); getchar(); } }
	//cutilSafeCall(hipMemcpy(rRot, state.d_pRot, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));
	//cutilSafeCall(hipMemcpy(rTrans, state.d_pTrans, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));
	//for (unsigned int i = 1; i < input.numberOfImages; i++) { if (isnan(rRot[i].x)) { printf("NaN in jtr pRot %d\n", i); getchar(); } }
	//for (unsigned int i = 1; i < input.numberOfImages; i++) { if (isnan(rTrans[i].x)) { printf("NaN in jtr pTrans %d\n", i); getchar(); } }

	PCGInit_Kernel2 << <blocksPerGrid, THREADS_PER_BLOCK >> >(N, state);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

	if (timer) timer->endEvent();

	//float scanAlpha;
	//cutilSafeCall(hipMemcpy(&scanAlpha, state.d_scanAlpha, sizeof(float), hipMemcpyDeviceToHost));
	//if (rRot) delete[] rRot;
	//if (rTrans) delete[] rTrans;
}

/////////////////////////////////////////////////////////////////////////
// PCG Iteration Parts
/////////////////////////////////////////////////////////////////////////

//inefficient
__global__ void PCGStep_Kernel_Dense_Brute(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfImages;							// Number of block variables
	const unsigned int x = blockIdx.x;

	if (x > 0 && x < N)
	{
		float3 rot, trans;
		applyJTJDenseBruteDevice(x, state, state.d_denseJtJ, input.numberOfImages, rot, trans); // A x p_k  => J^T x J x p_k 

		state.d_Ap_XRot[x] += rot;
		state.d_Ap_XTrans[x] += trans;
	}
}
__global__ void PCGStep_Kernel_Dense(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfImages;							// Number of block variables
	const unsigned int x = blockIdx.x;
	const unsigned int lane = threadIdx.x % WARP_SIZE;

	if (x > 0 && x < N)
	{
		float3 rot, trans;
		applyJTJDenseDevice(x, state, state.d_denseJtJ, input.numberOfImages, rot, trans, threadIdx.x);			// A x p_k  => J^T x J x p_k 

		if (lane == 0)
		{
			atomicAdd(&state.d_Ap_XRot[x].x, rot.x);
			atomicAdd(&state.d_Ap_XRot[x].y, rot.y);
			atomicAdd(&state.d_Ap_XRot[x].z, rot.z);

			atomicAdd(&state.d_Ap_XTrans[x].x, trans.x);
			atomicAdd(&state.d_Ap_XTrans[x].y, trans.y);
			atomicAdd(&state.d_Ap_XTrans[x].z, trans.z);
		}
	}
}

__global__ void PCGStep_Kernel0(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfCorrespondences;					// Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < N)
	{
		const float3 tmp = applyJDevice(x, input, state, parameters);		// A x p_k  => J^T x J x p_k 
		state.d_Jp[x] = tmp;												// store for next kernel call
	}
}

__global__ void PCGStep_Kernel1a(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfImages;							// Number of block variables
	const unsigned int x = blockIdx.x;
	const unsigned int lane = threadIdx.x % WARP_SIZE;

	if (x > 0 && x < N)
	{
		float3 rot, trans;
		applyJTDevice(x, input, state, parameters, rot, trans, threadIdx.x, lane);			// A x p_k  => J^T x J x p_k 

		if (lane == 0)
		{
			atomicAdd(&state.d_Ap_XRot[x].x, rot.x);
			atomicAdd(&state.d_Ap_XRot[x].y, rot.y);
			atomicAdd(&state.d_Ap_XRot[x].z, rot.z);

			atomicAdd(&state.d_Ap_XTrans[x].x, trans.x);
			atomicAdd(&state.d_Ap_XTrans[x].y, trans.y);
			atomicAdd(&state.d_Ap_XTrans[x].z, trans.z);
		}
	}
}

__global__ void PCGStep_Kernel1b(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfImages;								// Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	float d = 0.0f;
	if (x > 0 && x < N)
	{
		d = dot(state.d_pRot[x], state.d_Ap_XRot[x]) + dot(state.d_pTrans[x], state.d_Ap_XTrans[x]);		// x-th term of denominator of alpha
	}

	d = warpReduce(d);
	if (threadIdx.x % WARP_SIZE == 0)
	{
		atomicAdd(state.d_scanAlpha, d);
	}
}

__global__ void PCGStep_Kernel2(SolverInput input, SolverState state)
{
	const unsigned int N = input.numberOfImages;
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	const float dotProduct = state.d_scanAlpha[0];

	float b = 0.0f;
	if (x > 0 && x < N)
	{
		float alpha = 0.0f;
		if (dotProduct > FLOAT_EPSILON) alpha = state.d_rDotzOld[x] / dotProduct;		// update step size alpha

		state.d_deltaRot[x] = state.d_deltaRot[x] + alpha*state.d_pRot[x];			// do a decent step
		state.d_deltaTrans[x] = state.d_deltaTrans[x] + alpha*state.d_pTrans[x];	// do a decent step

		float3 rRot = state.d_rRot[x] - alpha*state.d_Ap_XRot[x];					// update residuum
		state.d_rRot[x] = rRot;														// store for next kernel call

		float3 rTrans = state.d_rTrans[x] - alpha*state.d_Ap_XTrans[x];				// update residuum
		state.d_rTrans[x] = rTrans;													// store for next kernel call

		float3 zRot = state.d_precondionerRot[x] * rRot;							// apply preconditioner M^-1
		state.d_zRot[x] = zRot;														// save for next kernel call

		float3 zTrans = state.d_precondionerTrans[x] * rTrans;						// apply preconditioner M^-1
		state.d_zTrans[x] = zTrans;													// save for next kernel call

		b = dot(zRot, rRot) + dot(zTrans, rTrans);									// compute x-th term of the nominator of beta
	}
	b = warpReduce(b);
	if (threadIdx.x % WARP_SIZE == 0)
	{
		atomicAdd(&state.d_scanAlpha[1], b);
	}
}

template<bool lastIteration>
__global__ void PCGStep_Kernel3(SolverInput input, SolverState state)
{
	const unsigned int N = input.numberOfImages;
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x > 0 && x < N)
	{
		const float rDotzNew = state.d_scanAlpha[1];
               								// get new nominator
		const float rDotzOld = state.d_rDotzOld[x];								// get old denominator

		float beta = 0.0f;
		if (rDotzOld > FLOAT_EPSILON) beta = rDotzNew / rDotzOld;				// update step size beta

		state.d_rDotzOld[x] = rDotzNew;											// save new rDotz for next iteration
		state.d_pRot[x] = state.d_zRot[x] + beta*state.d_pRot[x];		// update decent direction
		state.d_pTrans[x] = state.d_zTrans[x] + beta*state.d_pTrans[x];		// update decent direction


		state.d_Ap_XRot[x] = make_float3(0.0f, 0.0f, 0.0f);
		state.d_Ap_XTrans[x] = make_float3(0.0f, 0.0f, 0.0f);

		if (lastIteration)
		{
			//if (input.d_validImages[x]) { //not really necessary
#ifdef USE_LIE_SPACE //TODO just keep that matrix transforms around
			float3 rot, trans;
			computeLieUpdate(state.d_deltaRot[x], state.d_deltaTrans[x], state.d_xRot[x], state.d_xTrans[x], rot, trans);
			state.d_xRot[x] = rot;
			state.d_xTrans[x] = trans;
#else
			state.d_xRot[x] = state.d_xRot[x] + state.d_deltaRot[x];
			state.d_xTrans[x] = state.d_xTrans[x] + state.d_deltaTrans[x];
#endif
			//}
		}
	}
}

template<bool useSparse, bool useDense>
bool PCGIteration(SolverInput& input, SolverState& state, SolverParameters& parameters, SolverStateAnalysis& analysis, bool lastIteration, CUDATimer *timer)
{
	const unsigned int N = input.numberOfImages;	// Number of block variables
        // Do PCG step
	const int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	if (blocksPerGrid > THREADS_PER_BLOCK)
	{
		std::cout << "Too many variables for this block size. Maximum number of variables for two kernel scan: " << THREADS_PER_BLOCK*THREADS_PER_BLOCK << std::endl;
		while (1);
	}
	if (timer) timer->startEvent("PCGIteration");

	cutilSafeCall(hipMemset(state.d_scanAlpha, 0, sizeof(float) * 2));

	// sparse part
	if (useSparse) {
		const unsigned int Ncorr = input.numberOfCorrespondences;
		const int blocksPerGridCorr = (Ncorr + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
		PCGStep_Kernel0 << <blocksPerGridCorr, THREADS_PER_BLOCK >> >(input, state, parameters);
#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
#endif
		PCGStep_Kernel1a << < N, THREADS_PER_BLOCK_JT >> >(input, state, parameters);
#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
#endif
	}
	if (useDense) {
		//if (timer) timer->startEvent("apply JTJ dense");
		PCGStep_Kernel_Dense << < N, THREADS_PER_BLOCK_JT_DENSE >> >(input, state, parameters);
		//PCGStep_Kernel_Dense_Brute << < N, 1 >> >(input, state, parameters);
#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
#endif
		//if (timer) timer->endEvent();
	}
	//!!!debugging
	//float3* Ap_Rot = new float3[input.numberOfImages];
	//float3* Ap_Trans = new float3[input.numberOfImages];
	//cutilSafeCall(hipMemcpy(Ap_Rot, state.d_Ap_XRot, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));
	//cutilSafeCall(hipMemcpy(Ap_Trans, state.d_Ap_XTrans, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));
	//for (unsigned int i = 1; i < input.maxNumberOfImages; i++) { if (isnan(Ap_Rot[i].x)) { printf("NaN at Ap rot %d\n", i); getchar(); } }
	//for (unsigned int i = 1; i < input.maxNumberOfImages; i++) { if (isnan(Ap_Trans[i].x)) { printf("NaN at Ap trans %d\n", i); getchar(); } }
	//if (Ap_Rot) delete[] Ap_Rot;
	//if (Ap_Trans) delete[] Ap_Trans;
	//!!!debugging

	PCGStep_Kernel1b << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state, parameters);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

	PCGStep_Kernel2 << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
#ifdef ENABLE_EARLY_OUT //for convergence
	float scanAlpha; cutilSafeCall(hipMemcpy(&scanAlpha, state.d_scanAlpha, sizeof(float), hipMemcpyDeviceToHost));
	//if (fabs(scanAlpha) < 0.00005f) lastIteration = true;  //todo check this part
	//if (fabs(scanAlpha) < 1e-6) lastIteration = true;  //todo check this part
	if (fabs(scanAlpha) < 5e-7) { lastIteration = true; }  //todo check this part
#endif
	if (lastIteration) {
		PCGStep_Kernel3<true> << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state);
	}
	else {
		PCGStep_Kernel3<false> << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state);
	}

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	if (timer) timer->endEvent();

	return lastIteration;
}

#ifdef USE_LIE_SPACE //TODO
////////////////////////////////////////////////////////////////////
// matrix <-> pose
////////////////////////////////////////////////////////////////////
__global__ void convertLiePosesToMatricesCU_Kernel(const float3* d_rot, const float3* d_trans, unsigned int numTransforms, float4x4* d_transforms, float4x4* d_transformInvs)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < numTransforms) {
		poseToMatrix(d_rot[idx], d_trans[idx], d_transforms[idx]);
		d_transformInvs[idx] = d_transforms[idx].getInverse();
	}
}
extern "C"
void convertLiePosesToMatricesCU(const float3* d_rot, const float3* d_trans, unsigned int numTransforms, float4x4* d_transforms, float4x4* d_transformInvs)
{
	convertLiePosesToMatricesCU_Kernel << <(numTransforms + 8 - 1) / 8, 8 >> >(d_rot, d_trans, numTransforms, d_transforms, d_transformInvs);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}
#endif

////////////////////////////////////////////////////////////////////
// Main GN Solver Loop
////////////////////////////////////////////////////////////////////

extern "C" void solveBundlingStub(SolverInput& input, SolverState& state, SolverParameters& parameters, SolverStateAnalysis& analysis, float* convergenceAnalysis, CUDATimer *timer)
{
	if (convergenceAnalysis) {
		float initialResidual = EvalResidual(input, state, parameters, timer);
		convergenceAnalysis[0] = initialResidual; // initial residual
	}

	//!!!DEBUGGING
#ifdef PRINT_RESIDUALS_SPARSE
	if (parameters.weightSparse > 0) {
		if (input.numberOfCorrespondences == 0) { printf("ERROR: %d correspondences\n", input.numberOfCorrespondences); getchar(); }
		float initialResidual = EvalResidual(input, state, parameters, timer);
		printf("initial sparse = %f*%f = %f\n", parameters.weightSparse, initialResidual / parameters.weightSparse, initialResidual);
	}
#endif
	//float3* xRot = new float3[input.numberOfImages];	//remember the delete!
	//float3* xTrans = new float3[input.numberOfImages];
	//timer = new CUDATimer();
	//static unsigned int totalLinIters = 0, numLin = 0, totalNonLinIters = 0, numNonLin = 0;
	//!!!DEBUGGING

	for (unsigned int nIter = 0; nIter < parameters.nNonLinearIterations; nIter++)
	{
		parameters.weightSparse = input.weightsSparse[nIter];
		parameters.weightDenseDepth = input.weightsDenseDepth[nIter];
		parameters.weightDenseColor = input.weightsDenseColor[nIter];
		parameters.useDense = (parameters.weightDenseDepth > 0 || parameters.weightDenseColor > 0);
#ifdef USE_LIE_SPACE
		convertLiePosesToMatricesCU(state.d_xRot, state.d_xTrans, input.numberOfImages, state.d_xTransforms, state.d_xTransformInverses);
#endif
		if (parameters.useDense) parameters.useDense = BuildDenseSystem(input, state, parameters, timer); //don't solve dense if no overlapping frames found
		Initialization(input, state, parameters, timer);

		if (parameters.weightSparse > 0.0f) {
			if (parameters.useDense) {
				for (unsigned int linIter = 0; linIter < parameters.nLinIterations; linIter++)
					if (PCGIteration<true, true>(input, state, parameters, analysis, linIter == parameters.nLinIterations - 1, timer)) { break; }
			}
			else {
				for (unsigned int linIter = 0; linIter < parameters.nLinIterations; linIter++)
					if (PCGIteration<true, false>(input, state, parameters, analysis, linIter == parameters.nLinIterations - 1, timer)) {
						//totalLinIters += (linIter+1); numLin++; 
						break;
					}
			}
		}
		else {
			for (unsigned int linIter = 0; linIter < parameters.nLinIterations; linIter++)
				if (PCGIteration<false, true>(input, state, parameters, analysis, linIter == parameters.nLinIterations - 1, timer)) break;
		}
		//!!!debugging
		//cutilSafeCall(hipMemcpy(xRot, state.d_xRot, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));
		//cutilSafeCall(hipMemcpy(xTrans, state.d_xTrans, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));
		//!!!debugging
#ifdef PRINT_RESIDUALS_SPARSE
		if (parameters.weightSparse > 0) {
			float residual = EvalResidual(input, state, parameters, timer);
			printf("[niter %d] weight * sparse = %f*%f = %f\t[#corr = %d]\n", nIter, parameters.weightSparse, residual / parameters.weightSparse, residual, input.numberOfCorrespondences);
		}
#endif
		if (convergenceAnalysis) {
			float residual = EvalResidual(input, state, parameters, timer);
			convergenceAnalysis[nIter + 1] = residual;
		}

		//if (timer) timer->evaluate(true);

#ifdef ENABLE_EARLY_OUT //convergence
		//if (nIter < parameters.nNonLinearIterations - 1 && EvalGNConvergence(input, state, analysis, timer) < 0.01f) { //!!! TODO CHECK HOW THESE GENERALIZE
		if (nIter < parameters.nNonLinearIterations - 1 && EvalGNConvergence(input, state, analysis, timer) < 0.005f) { //0.001?
		//if (nIter < parameters.nNonLinearIterations - 1 && EvalGNConvergence(input, state, analysis, timer) < 0.001f) { 
			//if (!parameters.useDense) { totalNonLinIters += (nIter+1); numNonLin++; }
			break;
		}
		//else if (!parameters.useDense && nIter == parameters.nNonLinearIterations - 1) { totalNonLinIters += (nIter+1); numNonLin++; }
#endif
		}
	//!!!debugging
	//if (xRot) delete[] xRot;
	//if (xTrans) delete[] xTrans;
	//if (timer) { timer->evaluate(true, false); delete timer; }
	//if (!parameters.useDense) { printf("mean #pcg its = %f\tmean #gn its = %f\n", (float)totalLinIters / (float)numLin, (float)totalNonLinIters / (float)numNonLin); } //just stats for global solve
	//!!!debugging
	}

////////////////////////////////////////////////////////////////////
// build variables to correspondences lookup
////////////////////////////////////////////////////////////////////

__global__ void BuildVariablesToCorrespondencesTableDevice(EntryJ* d_correspondences, unsigned int numberOfCorrespondences,
	unsigned int maxNumCorrespondencesPerImage, int* d_variablesToCorrespondences, int* d_numEntriesPerRow)
{
	const unsigned int N = numberOfCorrespondences; // Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < N) {
		EntryJ& corr = d_correspondences[x];
		if (corr.isValid()) {
			int offset0 = atomicAdd(&d_numEntriesPerRow[corr.imgIdx_i], 1); // may overflow - need to check when read
			int offset1 = atomicAdd(&d_numEntriesPerRow[corr.imgIdx_j], 1); // may overflow - need to check when read
			if (offset0 < maxNumCorrespondencesPerImage && offset1 < maxNumCorrespondencesPerImage)	{
				d_variablesToCorrespondences[corr.imgIdx_i * maxNumCorrespondencesPerImage + offset0] = x;
				d_variablesToCorrespondences[corr.imgIdx_j * maxNumCorrespondencesPerImage + offset1] = x;
			}
			else { //invalidate
				printf("EXCEEDED MAX NUM CORR PER IMAGE IN SOLVER, INVALIDATING %d(%d,%d) [%d,%d | %d]\n",
					x, corr.imgIdx_i, corr.imgIdx_j, offset0, offset1, maxNumCorrespondencesPerImage); //debugging
				corr.setInvalid(); //make sure j corresponds to jt
			}
		}
	}
}

extern "C" void buildVariablesToCorrespondencesTableCUDA(EntryJ* d_correspondences, unsigned int numberOfCorrespondences, unsigned int maxNumCorrespondencesPerImage, int* d_variablesToCorrespondences, int* d_numEntriesPerRow, CUDATimer* timer)
{
	const unsigned int N = numberOfCorrespondences;

	if (timer) timer->startEvent(__FUNCTION__);

	BuildVariablesToCorrespondencesTableDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(d_correspondences, numberOfCorrespondences, maxNumCorrespondencesPerImage, d_variablesToCorrespondences, d_numEntriesPerRow);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

	if (timer) timer->endEvent();
}
