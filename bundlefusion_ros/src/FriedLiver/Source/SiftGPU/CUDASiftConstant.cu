
#include "SiftCameraParams.h"

__constant__ SiftCameraParams c_siftCameraParams;

extern "C" void updateConstantSiftCameraParams(const SiftCameraParams& params) {
	
	size_t size;
	cutilSafeCall(hipGetSymbolSize(&size, HIP_SYMBOL(c_siftCameraParams)));
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_siftCameraParams), &params, size, 0, hipMemcpyHostToDevice));
	
#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

}