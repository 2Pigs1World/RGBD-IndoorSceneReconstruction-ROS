#include "hip/hip_runtime.h"

#include "CUDAImageUtil.h"

#include "mLibCuda.h"//from #include "mibCuda.h" to #include "mLibCuda.h"

#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)

template<class T> void CUDAImageUtil::copy(T* d_output, T* d_input, unsigned int width, unsigned int height) {
	MLIB_CUDA_SAFE_CALL(hipMemcpy(d_output, d_input, sizeof(T)*width*height, hipMemcpyDeviceToDevice));
}

template<> void CUDAImageUtil::copy<float>(float* d_output, float* d_input, unsigned int width, unsigned int height){MLIB_CUDA_SAFE_CALL(hipMemcpy(d_output, d_input, sizeof(float)*width*height, hipMemcpyDeviceToDevice));}//change by guan
template<> void CUDAImageUtil::copy<uchar4>(uchar4* d_output, uchar4* d_input, unsigned int width, unsigned int height){MLIB_CUDA_SAFE_CALL(hipMemcpy(d_output, d_input, sizeof(uchar4)*width*height, hipMemcpyDeviceToDevice));}//change by guan



////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Resample Float Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

inline __device__ float bilinearInterpolationFloat(float x, float y, const float* d_input, unsigned int imageWidth, unsigned int imageHeight)
{
	const int2 p00 = make_int2(floor(x), floor(y));
	const int2 p01 = p00 + make_int2(0.0f, 1.0f);
	const int2 p10 = p00 + make_int2(1.0f, 0.0f);
	const int2 p11 = p00 + make_int2(1.0f, 1.0f);

	const float alpha = x - p00.x;
	const float beta = y - p00.y;

	float s0 = 0.0f; float w0 = 0.0f;
	if (p00.x < imageWidth && p00.y < imageHeight) { float v00 = d_input[p00.y*imageWidth + p00.x]; if (v00 != MINF) { s0 += (1.0f - alpha)*v00; w0 += (1.0f - alpha); } }
	if (p10.x < imageWidth && p10.y < imageHeight) { float v10 = d_input[p10.y*imageWidth + p10.x]; if (v10 != MINF) { s0 += alpha *v10; w0 += alpha; } }

	float s1 = 0.0f; float w1 = 0.0f;
	if (p01.x < imageWidth && p01.y < imageHeight) { float v01 = d_input[p01.y*imageWidth + p01.x]; if (v01 != MINF) { s1 += (1.0f - alpha)*v01; w1 += (1.0f - alpha); } }
	if (p11.x < imageWidth && p11.y < imageHeight) { float v11 = d_input[p11.y*imageWidth + p11.x]; if (v11 != MINF) { s1 += alpha *v11; w1 += alpha; } }

	const float p0 = s0 / w0;
	const float p1 = s1 / w1;

	float ss = 0.0f; float ww = 0.0f;
	if (w0 > 0.0f) { ss += (1.0f - beta)*p0; ww += (1.0f - beta); }
	if (w1 > 0.0f) { ss += beta *p1; ww += beta; }

	if (ww > 0.0f) return ss / ww;
	else		  return MINF;
}

//template<class T>
//__global__ void resample_Kernel(T* d_output, T* d_input, unsigned int inputWidth, unsigned int inputHeight, unsigned int outputWidth, unsigned int outputHeight)
//{
//	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
//	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
//
//	if (x < outputWidth && y < outputHeight)
//	{
//		const float scaleWidth = (float)(inputWidth - 1) / (float)(outputWidth - 1);
//		const float scaleHeight = (float)(inputHeight - 1) / (float)(outputHeight - 1);
//
//		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
//		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);
//
//		if (xInput < inputWidth && yInput < inputHeight)
//		{
//			if (std::is_same<T, float>::value) {
//				d_output[y*outputWidth + x] = (T)bilinearInterpolationFloat(x*scaleWidth, y*scaleHeight, (float*)d_input, inputWidth, inputHeight);
//			}
//			else if (std::is_same<T, uchar4>::value) {
//				d_output[y*outputWidth + x] = d_input[yInput*inputWidth + xInput];
//			}
//			else {
//				//static_assert(false, "bla");
//			}
//		}
//	}
//}
//
//template<class T> void CUDAImageUtil::resample(T* d_output, unsigned int outputWidth, unsigned int outputHeight, T* d_input, unsigned int inputWidth, unsigned int inputHeight) {
//
//	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
//	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);
//
//	resample_Kernel << <gridSize, blockSize >> >(d_output, d_input, inputWidth, inputHeight, outputWidth, outputHeight);
//
//#ifdef _DEBUG
//	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
//	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
//#endif
//}


__global__ void resampleFloat_Kernel(float* d_output, unsigned int outputWidth, unsigned int outputHeight, const float* d_input, unsigned int inputWidth, unsigned int inputHeight)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < outputWidth && y < outputHeight)
	{
		const float scaleWidth = (float)(inputWidth-1) / (float)(outputWidth-1);
		const float scaleHeight = (float)(inputHeight-1) / (float)(outputHeight-1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

		if (xInput < inputWidth && yInput < inputHeight) {
			d_output[y*outputWidth + x] = d_input[yInput*inputWidth + xInput];
			//d_output[y*outputWidth + x] = bilinearInterpolationFloat(x*scaleWidth, y*scaleHeight, d_input, inputWidth, inputHeight);
		}
	}
}

void CUDAImageUtil::resampleFloat(float* d_output, unsigned int outputWidth, unsigned int outputHeight, const float* d_input, unsigned int inputWidth, unsigned int inputHeight) {

	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleFloat_Kernel << <gridSize, blockSize >> >(d_output, outputWidth, outputHeight, d_input, inputWidth, inputHeight);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}

__global__ void resampleFloat4_Kernel(float4* d_output, unsigned int outputWidth, unsigned int outputHeight, const float4* d_input, unsigned int inputWidth, unsigned int inputHeight)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < outputWidth && y < outputHeight)
	{
		const float scaleWidth = (float)(inputWidth-1) / (float)(outputWidth-1);
		const float scaleHeight = (float)(inputHeight-1) / (float)(outputHeight-1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

		if (xInput < inputWidth && yInput < inputHeight) {
			d_output[y*outputWidth + x] = d_input[yInput*inputWidth + xInput];
			//d_output[y*outputWidth + x] = bilinearInterpolationFloat(x*scaleWidth, y*scaleHeight, d_input, inputWidth, inputHeight);
		}
	}
}
void CUDAImageUtil::resampleFloat4(float4* d_output, unsigned int outputWidth, unsigned int outputHeight, const float4* d_input, unsigned int inputWidth, unsigned int inputHeight) {

	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleFloat4_Kernel << <gridSize, blockSize >> >(d_output, outputWidth, outputHeight, d_input, inputWidth, inputHeight);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}



__global__ void resampleUCHAR4_Kernel(uchar4* d_output, unsigned int outputWidth, unsigned int outputHeight, const uchar4* d_input, unsigned int inputWidth, unsigned int inputHeight)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < outputWidth && y < outputHeight)
	{
		const float scaleWidth = (float)(inputWidth-1) / (float)(outputWidth-1);
		const float scaleHeight = (float)(inputHeight-1) / (float)(outputHeight-1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

		if (xInput < inputWidth && yInput < inputHeight) {
			d_output[y*outputWidth + x] = d_input[yInput*inputWidth + xInput];
		}
	}
}

void CUDAImageUtil::resampleUCHAR4(uchar4* d_output, unsigned int outputWidth, unsigned int outputHeight, const uchar4* d_input, unsigned int inputWidth, unsigned int inputHeight) {

	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleUCHAR4_Kernel << <gridSize, blockSize >> >(d_output, outputWidth, outputHeight, d_input, inputWidth, inputHeight);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Color to Intensity
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__host__ __device__
float convertToIntensity(const uchar4& c) {
	return (0.299f*c.x + 0.587f*c.y + 0.114f*c.z) / 255.0f;
}

__global__ void convertUCHAR4ToIntensityFloat_Kernel(float* d_output, const uchar4* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		d_output[y*width + x] = convertToIntensity(d_input[y*width + x]);
	}
}

void CUDAImageUtil::convertUCHAR4ToIntensityFloat(float* d_output, const uchar4* d_input, unsigned int width, unsigned int height) {

	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertUCHAR4ToIntensityFloat_Kernel << <gridSize, blockSize >> >(d_output, d_input, width, height);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}

__global__ void resampleToIntensity_Kernel(float* d_output, unsigned int outputWidth, unsigned int outputHeight, const uchar4* d_input, unsigned int inputWidth, unsigned int inputHeight)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < outputWidth && y < outputHeight)
	{
		const float scaleWidth = (float)(inputWidth-1) / (float)(outputWidth-1);
		const float scaleHeight = (float)(inputHeight-1) / (float)(outputHeight-1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

		if (xInput < inputWidth && yInput < inputHeight) {
			d_output[y*outputWidth + x] = convertToIntensity(d_input[yInput*inputWidth + xInput]);
		}
	}
}

void CUDAImageUtil::resampleToIntensity(float* d_output, unsigned int outputWidth, unsigned int outputHeight, const uchar4* d_input, unsigned int inputWidth, unsigned int inputHeight) {

	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleToIntensity_Kernel << <gridSize, blockSize >> >(d_output, outputWidth, outputHeight, d_input, inputWidth, inputHeight);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// derivatives 
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void computeIntensityDerivatives_Kernel(float2* d_output, const float* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < width && y < height)
	{
		d_output[y*width + x] = make_float2(MINF, MINF);

		//derivative
		if (x > 0 && x < width - 1 && y > 0 && y < height - 1)
		{ 
			float pos00 = d_input[(y - 1)*width + (x - 1)]; if (pos00 == MINF) return;
			float pos01 = d_input[(y - 0)*width + (x - 1)];	if (pos01 == MINF) return;
			float pos02 = d_input[(y + 1)*width + (x - 1)];	if (pos02 == MINF) return;

			float pos10 = d_input[(y - 1)*width + (x - 0)]; if (pos10 == MINF) return;
			//float pos11 = d_input[(y-0)*width + (x-0)]; if (pos11 == MINF) return;
			float pos12 = d_input[(y + 1)*width + (x - 0)]; if (pos12 == MINF) return;

			float pos20 = d_input[(y - 1)*width + (x + 1)]; if (pos20 == MINF) return;
			float pos21 = d_input[(y - 0)*width + (x + 1)]; if (pos21 == MINF) return;
			float pos22 = d_input[(y + 1)*width + (x + 1)]; if (pos22 == MINF) return;

			float resU = (-1.0f)*pos00 + (1.0f)*pos20 +
				(-2.0f)*pos01 + (2.0f)*pos21 +
				(-1.0f)*pos02 + (1.0f)*pos22;
			resU /= 8.0f;

			float resV = (-1.0f)*pos00 + (-2.0f)*pos10 + (-1.0f)*pos20 +
				(1.0f)*pos02 + (2.0f)*pos12 + (1.0f)*pos22;
			resV /= 8.0f;

			d_output[y*width + x] = make_float2(resU, resV);
		}
	}
}

void CUDAImageUtil::computeIntensityDerivatives(float2* d_output, const float* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeIntensityDerivatives_Kernel << <gridSize, blockSize >> >(d_output, d_input, width, height);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}

__global__ void computeIntensityGradientMagnitude_Kernel(float* d_output, const float* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < width && y < height)
	{
		d_output[y*width + x] = MINF;

		//derivative
		if (x > 0 && x < width - 1 && y > 0 && y < height - 1)
		{ 
			float pos00 = d_input[(y - 1)*width + (x - 1)]; if (pos00 == MINF) return;
			float pos01 = d_input[(y - 0)*width + (x - 1)];	if (pos01 == MINF) return;
			float pos02 = d_input[(y + 1)*width + (x - 1)];	if (pos02 == MINF) return;

			float pos10 = d_input[(y - 1)*width + (x - 0)]; if (pos10 == MINF) return;
			//float pos11 = d_input[(y-0)*width + (x-0)]; if (pos11 == MINF) return;
			float pos12 = d_input[(y + 1)*width + (x - 0)]; if (pos12 == MINF) return;

			float pos20 = d_input[(y - 1)*width + (x + 1)]; if (pos20 == MINF) return;
			float pos21 = d_input[(y - 0)*width + (x + 1)]; if (pos21 == MINF) return;
			float pos22 = d_input[(y + 1)*width + (x + 1)]; if (pos22 == MINF) return;

			float resU = (-1.0f)*pos00 + (1.0f)*pos20 +
				(-2.0f)*pos01 + (2.0f)*pos21 +
				(-1.0f)*pos02 + (1.0f)*pos22;
			//resU /= 8.0f;

			float resV = (-1.0f)*pos00 + (-2.0f)*pos10 + (-1.0f)*pos20 +
				(1.0f)*pos02 + (2.0f)*pos12 + (1.0f)*pos22;
			//resV /= 8.0f;

			d_output[y*width + x] = sqrt(resU * resU + resV * resV);
		}
	}
}
void CUDAImageUtil::computeIntensityGradientMagnitude(float* d_output, const float* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeIntensityGradientMagnitude_Kernel << <gridSize, blockSize >> >(d_output, d_input, width, height);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}



////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Convert Depth to Camera Space Positions
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void convertDepthFloatToCameraSpaceFloat4_Kernel(float4* d_output, const float* d_input, float4x4 intrinsicsInv, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		d_output[y*width + x] = make_float4(MINF, MINF, MINF, MINF);

		float depth = d_input[y*width + x];

		if (depth != MINF)
		{
			float4 cameraSpace(intrinsicsInv*make_float4((float)x*depth, (float)y*depth, depth, depth));
			d_output[y*width + x] = make_float4(cameraSpace.x, cameraSpace.y, cameraSpace.w, 1.0f);
			//d_output[y*width + x] = make_float4(depthCameraData.kinectDepthToSkeleton(x, y, depth), 1.0f);
		}
	}
}

void CUDAImageUtil::convertDepthFloatToCameraSpaceFloat4(float4* d_output, const float* d_input, const float4x4& intrinsicsInv, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertDepthFloatToCameraSpaceFloat4_Kernel << <gridSize, blockSize >> >(d_output, d_input, intrinsicsInv, width, height);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Compute Normal Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void computeNormals_Kernel(float4* d_output, const float4* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	d_output[y*width + x] = make_float4(MINF, MINF, MINF, MINF);

	if (x > 0 && x < width - 1 && y > 0 && y < height - 1)
	{
		const float4 CC = d_input[(y + 0)*width + (x + 0)];
		const float4 PC = d_input[(y + 1)*width + (x + 0)];
		const float4 CP = d_input[(y + 0)*width + (x + 1)];
		const float4 MC = d_input[(y - 1)*width + (x + 0)];
		const float4 CM = d_input[(y + 0)*width + (x - 1)];

		if (CC.x != MINF && PC.x != MINF && CP.x != MINF && MC.x != MINF && CM.x != MINF)
		{
			const float3 n = cross(make_float3(PC) - make_float3(MC), make_float3(CP) - make_float3(CM));
			const float  l = length(n);

			if (l > 0.0f)
			{
				d_output[y*width + x] = make_float4(n / -l, 0.0f);
			}
		}
	}
}

void CUDAImageUtil::computeNormals(float4* d_output, const float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeNormals_Kernel << <gridSize, blockSize >> >(d_output, d_input, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

__global__ void computeNormalsSobel_Kernel(float4* d_output, const float4* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	d_output[y*width + x] = make_float4(MINF, MINF, MINF, MINF);

	if (x > 0 && x < width - 1 && y > 0 && y < height - 1)
	{
		float4 pos00 = d_input[(y - 1)*width + (x - 1)]; if (pos00.x == MINF) return;
		float4 pos01 = d_input[(y - 0)*width + (x - 1)]; if (pos01.x == MINF) return;
		float4 pos02 = d_input[(y + 1)*width + (x - 1)]; if (pos02.x == MINF) return;

		float4 pos10 = d_input[(y - 1)*width + (x - 0)]; if (pos10.x == MINF) return;
		//float4 pos11 = d_input[(y-0)*width + (x-0)]; if (pos11.x == MINF) return;
		float4 pos12 = d_input[(y + 1)*width + (x - 0)]; if (pos12.x == MINF) return;

		float4 pos20 = d_input[(y - 1)*width + (x + 1)]; if (pos20.x == MINF) return;
		float4 pos21 = d_input[(y - 0)*width + (x + 1)]; if (pos21.x == MINF) return;
		float4 pos22 = d_input[(y + 1)*width + (x + 1)]; if (pos22.x == MINF) return;

		float4 resU = (-1.0f)*pos00 + (1.0f)*pos20 +
			(-2.0f)*pos01 + (2.0f)*pos21 +
			(-1.0f)*pos02 + (1.0f)*pos22;

		float4 resV = (-1.0f)*pos00 + (-2.0f)*pos10 + (-1.0f)*pos20 +
			(1.0f)*pos02 + (2.0f)*pos12 + (1.0f)*pos22;

		const float3 n = cross(make_float3(resU.x, resU.y, resU.z), make_float3(resV.x, resV.y, resV.z));
		const float  l = length(n);

		if (l > 0.0f) d_output[y*width + x] = make_float4(n / l, 0.0f);
	}
}

void CUDAImageUtil::computeNormalsSobel(float4* d_output, const float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeNormalsSobel_Kernel << <gridSize, blockSize >> >(d_output, d_input, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

__global__ void convertNormalsFloat4ToUCHAR4_Kernel(uchar4* d_output, const float4* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		d_output[y*width + x] = make_uchar4(0, 0, 0, 0);

		float4 p = d_input[y*width + x];

		if (p.x != MINF)
		{
			p = (p + 1.0f) / 2.0f; // -> [0, 1]
			d_output[y*width + x] = make_uchar4((uchar)round(p.x * 255), (uchar)round(p.y * 255), (uchar)round(p.z * 255), 0);
		}
	}
}

void CUDAImageUtil::convertNormalsFloat4ToUCHAR4(uchar4* d_output, const float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertNormalsFloat4ToUCHAR4_Kernel << <gridSize, blockSize >> >(d_output, d_input, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Joint Bilateral Filter
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

inline __device__ float gaussD(float sigma, int x, int y)
{
	return exp(-((x*x + y*y) / (2.0f*sigma*sigma)));
}
inline __device__ float gaussR(float sigma, float dist)
{
	return exp(-(dist*dist) / (2.0*sigma*sigma));
}

__global__ void bilateralFilterUCHAR4_Kernel(uchar4* d_output, uchar4* d_color, float* d_depth, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	d_output[y*width + x] = d_color[y*width + x];

	float3 sum = make_float3(0.0f, 0.0f, 0.0f);
	float sumWeight = 0.0f;

	const float depthCenter = d_depth[y*width + x];
	if (depthCenter != MINF)
	{
		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const uchar4 cur = d_color[n*width + m];
					const float currentDepth = d_depth[n*width + m];

					if (currentDepth != MINF) {
						const float weight = gaussD(sigmaD, m - x, n - y)*gaussR(sigmaR, currentDepth - depthCenter);

						sumWeight += weight;
						sum += weight*make_float3(cur.x, cur.y, cur.z);
					}
				}
			}
		}

		if (sumWeight > 0.0f) {
			float3 res = sum / sumWeight;
			d_output[y*width + x] = make_uchar4((uchar)res.x, (uchar)res.y, (uchar)res.z, 255);
		}
	}
}

void CUDAImageUtil::jointBilateralFilterColorUCHAR4(uchar4* d_output, uchar4* d_input, float* d_depth, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	bilateralFilterUCHAR4_Kernel << <gridSize, blockSize >> >(d_output, d_input, d_depth, sigmaD, sigmaR, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

__global__ void bilateralFilterFloat_Kernel(float* d_output, float* d_input, float* d_depth, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	d_output[y*width + x] = MINF;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	const float depthCenter = d_depth[y*width + x];
	if (depthCenter != MINF)
	{
		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const float cur = d_input[n*width + m];
					const float currentDepth = d_depth[n*width + m];

					if (currentDepth != MINF && fabs(depthCenter - currentDepth) < sigmaR)
					{ //const float weight = gaussD(sigmaD, m - x, n - y)*gaussR(sigmaR, currentDepth - depthCenter);
						const float weight = gaussD(sigmaD, m - x, n - y);
						sumWeight += weight;
						sum += weight*cur;
					}
				}
			}
		}

		if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
	}
}
void CUDAImageUtil::jointBilateralFilterFloat(float* d_output, float* d_input, float* d_depth, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	bilateralFilterFloat_Kernel << <gridSize, blockSize >> >(d_output, d_input, d_depth, sigmaD, sigmaR, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

__global__ void adaptiveBilateralFilterIntensity_Kernel(float* d_output, const float* d_input, const float* d_depth, float sigmaD, float sigmaR, float adaptFactor, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	d_output[y*width + x] = MINF;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	const float depthCenter = d_depth[y*width + x];
	if (depthCenter != MINF)
	{
		const float curSigma = sigmaD * adaptFactor / depthCenter;
		const int kernelRadius = (int)ceil(2.0*curSigma);

		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const float cur = d_input[n*width + m];
					const float currentDepth = d_depth[n*width + m];

					if (currentDepth != MINF && fabs(depthCenter - currentDepth) < sigmaR)
					{ //const float weight = gaussD(curSigma, m - x, n - y)*gaussR(sigmaR, currentDepth - depthCenter);
						const float weight = gaussD(curSigma, m - x, n - y);
						sumWeight += weight;
						sum += weight*cur;
					}
				}
			}
		}

		if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
	}
}
void CUDAImageUtil::adaptiveBilateralFilterIntensity(float* d_output, const float* d_input, const float* d_depth, float sigmaD, float sigmaR, float adaptFactor, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	adaptiveBilateralFilterIntensity_Kernel << <gridSize, blockSize >> >(d_output, d_input, d_depth, sigmaD, sigmaR, adaptFactor, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Erode Depth Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void erodeDepthMapDevice(float* d_output, float* d_input, int structureSize, int width, int height, float dThresh, float fracReq)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;


	if (x >= 0 && x < width && y >= 0 && y < height)
	{


		unsigned int count = 0;

		float oldDepth = d_input[y*width + x];
		for (int i = -structureSize; i <= structureSize; i++)
		{
			for (int j = -structureSize; j <= structureSize; j++)
			{
				if (x + j >= 0 && x + j < width && y + i >= 0 && y + i < height)
				{
					float depth = d_input[(y + i)*width + (x + j)];
					if (depth == MINF || depth == 0.0f || fabs(depth - oldDepth) > dThresh)
					{
						count++;
						//d_output[y*width+x] = MINF;
						//return;
					}
				}
			}
		}

		unsigned int sum = (2 * structureSize + 1)*(2 * structureSize + 1);
		if ((float)count / (float)sum >= fracReq) {
			d_output[y*width + x] = MINF;
		}
		else {
			d_output[y*width + x] = d_input[y*width + x];
		}
	}
}

void CUDAImageUtil::erodeDepthMap(float* d_output, float* d_input, int structureSize, unsigned int width, unsigned int height, float dThresh, float fracReq)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	erodeDepthMapDevice << <gridSize, blockSize >> >(d_output, d_input, structureSize, width, height, dThresh, fracReq);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}



////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Gauss Filter Float Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void gaussFilterDepthMapDevice(float* d_output, const float* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	d_output[y*width + x] = MINF;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	const float depthCenter = d_input[y*width + x];
	if (depthCenter != MINF)
	{
		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const float currentDepth = d_input[n*width + m];

					if (currentDepth != MINF && fabs(depthCenter - currentDepth) < sigmaR)
					{
						const float weight = gaussD(sigmaD, m - x, n - y);

						sumWeight += weight;
						sum += weight*currentDepth;
					}
				}
			}
		}
	}

	if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
}

void CUDAImageUtil::gaussFilterDepthMap(float* d_output, const float* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	gaussFilterDepthMapDevice << <gridSize, blockSize >> >(d_output, d_input, sigmaD, sigmaR, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

__global__ void gaussFilterIntensityDevice(float* d_output, const float* d_input, float sigmaD, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	//d_output[y*width + x] = MINF;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	//const float center = d_input[y*width + x];
	//if (center != MINF) {
	for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
	{
		for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
		{
			if (m >= 0 && n >= 0 && m < width && n < height)
			{
				const float current = d_input[n*width + m];

				//if (current != MINF && fabs(center - current) < sigmaR) {
				const float weight = gaussD(sigmaD, m - x, n - y);

				sumWeight += weight;
				sum += weight*current;
				//}
			}
		}
	}
	//}

	if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
}

void CUDAImageUtil::gaussFilterIntensity(float* d_output, const float* d_input, float sigmaD, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	gaussFilterIntensityDevice << <gridSize, blockSize >> >(d_output, d_input, sigmaD, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// adaptive gauss filter float map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void adaptiveGaussFilterDepthMap_Kernel(float* d_output, const float* d_input, float sigmaD, float sigmaR,
	unsigned int width, unsigned int height, float adaptFactor)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;


	d_output[y*width + x] = MINF;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	const float depthCenter = d_input[y*width + x];
	if (depthCenter != MINF)
	{
		const float curSigma = sigmaD / depthCenter * adaptFactor;
		const int kernelRadius = (int)ceil(2.0*curSigma);

		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const float currentDepth = d_input[n*width + m];

					if (currentDepth != MINF && fabs(depthCenter - currentDepth) < sigmaR)
					{
						const float weight = gaussD(curSigma, m - x, n - y);

						sumWeight += weight;
						sum += weight*currentDepth;
					}
				}
			}
		}
	}

	if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
}
void CUDAImageUtil::adaptiveGaussFilterDepthMap(float* d_output, const float* d_input, float sigmaD, float sigmaR, float adaptFactor, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	adaptiveGaussFilterDepthMap_Kernel << <gridSize, blockSize >> >(d_output, d_input, sigmaD, sigmaR, width, height, adaptFactor);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

__global__ void adaptiveGaussFilterIntensity_Kernel(float* d_output, const float* d_input, const float* d_depth, float sigmaD,
	unsigned int width, unsigned int height, float adaptFactor)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	d_output[y*width + x] = MINF; //(should not be used in the case of no valid depth)

	const float depthCenter = d_depth[y*width + x];
	if (depthCenter != MINF)
	{
		const float curSigma = sigmaD / depthCenter * adaptFactor;
		const int kernelRadius = (int)ceil(2.0*curSigma);

		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const float currentDepth = d_depth[n*width + m];
					if (currentDepth != MINF) // && fabs(depthCenter - currentDepth) < sigmaR)
					{
						const float current = d_input[n*width + m];
						const float weight = gaussD(curSigma, m - x, n - y);

						sumWeight += weight;
						sum += weight*current;
					}
				}
			}
		}
	}

	if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
}
void CUDAImageUtil::adaptiveGaussFilterIntensity(float* d_output, const float* d_input, const float* d_depth, float sigmaD, float adaptFactor, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	adaptiveGaussFilterIntensity_Kernel << <gridSize, blockSize >> >(d_output, d_input, d_depth, sigmaD, width, height, adaptFactor);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}


