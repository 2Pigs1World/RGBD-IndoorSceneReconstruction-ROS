#include "hip/hip_runtime.h"
#ifndef _CAMERA_UTIL_
#define _CAMERA_UTIL_

#include <cutil_inline.h>
#include <cutil_math.h>

#include "cuda_SimpleMatrixUtil.h"
#include "DepthCameraUtil.h"

#ifndef BYTE
#define BYTE unsigned char
#endif

#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)

#ifndef HIP_PI_F
#define HIP_PI_F 3.141592654f
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Compute Copy Float Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void copyFloatMapDevice(float* d_output, float* d_input, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	d_output[y*width+x] = d_input[y*width+x];
}

extern "C" void copyFloatMap(float* d_output, float* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	copyFloatMapDevice<<<gridSize, blockSize>>>(d_output, d_input, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

__global__ void copyDepthFloatMapDevice(float* d_output, float* d_input, unsigned int width, unsigned int height, float minDepth, float maxDepth)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	const float depth = d_input[y*width+x];
	if (depth >= minDepth && depth <= maxDepth) 
		d_output[y*width+x] = depth;
	else
		d_output[y*width+x] = MINF;
}

extern "C" void copyDepthFloatMap(float* d_output, float* d_input, unsigned int width, unsigned int height, float minDepth, float maxDepth)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	copyDepthFloatMapDevice<<<gridSize, blockSize>>>(d_output, d_input, width, height,minDepth, maxDepth);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Compute Copy Float Map Fill
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void initializeOptimizerMapsDevice(float* d_output, float* d_input, float* d_input2, float* d_mask, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	const float depth = d_input[y*width+x];
	if(d_mask[y*width+x] != MINF) { d_output[y*width+x] = depth; }
	else						  { d_output[y*width+x] = MINF; d_input[y*width+x] = MINF; d_input2[y*width+x] = MINF; }
}

extern "C" void initializeOptimizerMaps(float* d_output, float* d_input, float* d_input2, float* d_mask, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	initializeOptimizerMapsDevice<<<gridSize, blockSize>>>(d_output, d_input, d_input2, d_mask, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Copy Float4 Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void copyFloat4MapDevice(float4* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	d_output[y*width+x] = d_input[y*width+x];
}

extern "C" void copyFloat4Map(float4* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	copyFloat4MapDevice<<<gridSize, blockSize>>>(d_output, d_input, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Convert Raw Color to float
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void convertColorRawToFloatDevice(float4* d_output, BYTE* d_input, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;
	
	uchar4 c = make_uchar4(d_input[4*(y*width+x)+0], d_input[4*(y*width+x)+1], d_input[4*(y*width+x)+2], d_input[4*(y*width+x)+3]);
	//if (c.x == 0 && c.y == 0 && c.z == 0) { // NO INVALID COLORS!
	//	d_output[y*width+x] = make_float4(MINF, MINF, MINF, MINF); 
	//} else {
		d_output[y*width+x] = make_float4(c.x/255.0f, c.y/255.0f, c.z/255.0f, c.w/255);	
	//}
}

extern "C" void convertColorRawToFloat4(float4* d_output, BYTE* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertColorRawToFloatDevice<<<gridSize, blockSize>>>(d_output, d_input, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Convert Float4 Color to UCHAR4
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void convertColorFloat4ToUCHAR4Device(uchar4* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	float4 color = d_input[y*width+x];
	d_output[y*width+x] = make_uchar4(color.x*255.0f, color.y*255.0f, color.z*255.0f, color.w*255.0f);	
}

extern "C" void convertColorFloat4ToUCHAR4(uchar4* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 blockSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 gridSize(T_PER_BLOCK, T_PER_BLOCK);

	convertColorFloat4ToUCHAR4Device<<<blockSize, gridSize>>>(d_output, d_input, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Mask Color Map using Depth
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void maskColorMapFloat4MapDevice(float4* d_inputColor, float4* d_inputDepth, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	float4 color = d_inputColor[y*width+x];
	
	if(d_inputDepth[y*width+x].x != MINF)	d_inputColor[y*width+x] = color;	
	else									d_inputColor[y*width+x] = make_float4(MINF, MINF, MINF, MINF);
}

extern "C" void maskColorMapFloat4Map(float4* d_inputColor, float4* d_inputDepth, unsigned int width, unsigned int height)
{
	const dim3 blockSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 gridSize(T_PER_BLOCK, T_PER_BLOCK);

	maskColorMapFloat4MapDevice<<<blockSize, gridSize>>>(d_inputColor, d_inputDepth, width, height);
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Convert Color to Intensity Float4
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void convertColorToIntensityFloat4Device(float4* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	const float4 color = d_input[y*width+x];
	const float I = 0.299f*color.x + 0.587f*color.y + 0.114f*color.z;

	d_output[y*width+x] = make_float4(I, I, I, 1.0f);
}

extern "C" void convertColorToIntensityFloat4(float4* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertColorToIntensityFloat4Device<<<gridSize, blockSize>>>(d_output, d_input, width, height);

	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Convert Color to Intensity
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void convertColorToIntensityFloatDevice(float* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	const float4 color = d_input[y*width+x];
	d_output[y*width+x] = 0.299f*color.x + 0.587f*color.y + 0.114f*color.z;
}

extern "C" void convertColorToIntensityFloat(float* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertColorToIntensityFloatDevice<<<gridSize, blockSize>>>(d_output, d_input, width, height);

	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Convert depth map to color map view
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void convertDepthToColorSpaceDevice(float* d_output, float* d_input, float4x4 depthIntrinsicsInv, float4x4 colorIntrinsics, float4x4 depthExtrinsicsInv, float4x4 colorExtrinsics, unsigned int depthWidth, unsigned int depthHeight, unsigned int colorWidth, unsigned int colorHeight)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x < depthWidth && y < depthHeight)
	{
		const float depth = d_input[y*depthWidth+x];

		if(depth != MINF && depth < 1.0f)
		{
			// Cam space depth
			float4 depthCamSpace = depthIntrinsicsInv*make_float4((float)x*depth, (float)y*depth, depth, depth);
			depthCamSpace = make_float4(depthCamSpace.x, depthCamSpace.y, depthCamSpace.w, 1.0f);

			// World Space
			const float4 worldSpace = depthExtrinsicsInv*depthCamSpace;

			// Cam space color
			float4 colorCamSpace = colorExtrinsics*worldSpace;
			//colorCamSpace = make_float4(colorCamSpace.x, colorCamSpace.y, 0.0f, colorCamSpace.z);
			colorCamSpace = make_float4(colorCamSpace.x, colorCamSpace.y, colorCamSpace.z, 1.0f);

			// Get coordinates in color image and set pixel to new depth
			const float4 screenSpaceColor  = colorIntrinsics*colorCamSpace;
			//const unsigned int cx = (unsigned int)(screenSpaceColor.x/screenSpaceColor.w + 0.5f);
			//const unsigned int cy = (unsigned int)(screenSpaceColor.y/screenSpaceColor.w + 0.5f);
			const unsigned int cx = (unsigned int)(screenSpaceColor.x/screenSpaceColor.z + 0.5f);
			const unsigned int cy = (unsigned int)(screenSpaceColor.y/screenSpaceColor.z + 0.5f);

			//if(cx < colorWidth && cy < colorHeight) d_output[cy*colorWidth+cx] = screenSpaceColor.w; // Check for minimum !!!
			if(cx < colorWidth && cy < colorHeight) d_output[cy*colorWidth+cx] = screenSpaceColor.z; // Check for minimum !!!
		}
	}
}

extern "C" void convertDepthToColorSpace(float* d_output, float* d_input, float4x4 depthIntrinsicsInv, float4x4 colorIntrinsics, float4x4 depthExtrinsicsInv, float4x4 colorExtrinsics, unsigned int depthWidth, unsigned int depthHeight, unsigned int colorWidth, unsigned int colorHeight)
{
	const dim3 gridSize((depthWidth + T_PER_BLOCK - 1)/T_PER_BLOCK, (depthHeight + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertDepthToColorSpaceDevice<<<gridSize, blockSize>>>(d_output, d_input, depthIntrinsicsInv, colorIntrinsics, depthExtrinsicsInv, colorExtrinsics, depthWidth, depthHeight, colorWidth, colorHeight);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Set invalid float map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void setInvalidFloatMapDevice(float* d_output, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	d_output[y*width+x] = MINF;
}

extern "C" void setInvalidFloatMap(float* d_output, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	setInvalidFloatMapDevice<<<gridSize, blockSize>>>(d_output, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Set invalid float4 map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void setInvalidFloat4MapDevice(float4* d_output, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	d_output[y*width+x] = make_float4(MINF, MINF, MINF ,MINF);
}

extern "C" void setInvalidFloat4Map(float4* d_output, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	setInvalidFloat4MapDevice<<<gridSize, blockSize>>>(d_output, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Convert Depth to Camera Space Positions
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void convertDepthFloatToCameraSpaceFloat4Device(float4* d_output, float* d_input, float4x4 intrinsicsInv, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		d_output[y*width+x] = make_float4(MINF, MINF, MINF, MINF);

		float depth = d_input[y*width+x];

		if(depth != MINF)
		{
			//float4 cameraSpace(intrinsicsInv*make_float4((float)x*depth, (float)y*depth, depth, depth));
			//d_output[y*width+x] = make_float4(cameraSpace.x, cameraSpace.y, cameraSpace.w, 1.0f);
			d_output[y*width + x] = make_float4(DepthCameraData::kinectDepthToSkeleton(x, y, depth), 1.0f);
		}
	}
}

extern "C" void convertDepthFloatToCameraSpaceFloat4(float4* d_output, float* d_input, float4x4 intrinsicsInv, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertDepthFloatToCameraSpaceFloat4Device<<<gridSize, blockSize>>>(d_output, d_input, intrinsicsInv, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Bilateral Filter Float Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

inline __device__ float gaussR(float sigma, float dist)
{
	return exp(-(dist*dist)/(2.0*sigma*sigma));
}

inline __device__ float linearR(float sigma, float dist)
{
	return max(1.0f, min(0.0f, 1.0f-(dist*dist)/(2.0*sigma*sigma)));
}

inline __device__ float gaussD(float sigma, int x, int y)
{
	return exp(-((x*x+y*y)/(2.0f*sigma*sigma)));
}

inline __device__ float gaussD(float sigma, int x)
{
	return exp(-((x*x)/(2.0f*sigma*sigma)));
}

__global__ void bilateralFilterFloatMapDevice(float* d_output, float* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	d_output[y*width+x] = MINF;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	const float depthCenter = d_input[y*width+x];
	if(depthCenter != MINF)
	{
		for(int m = x-kernelRadius; m <= x+kernelRadius; m++)
		{
			for(int n = y-kernelRadius; n <= y+kernelRadius; n++)
			{		
				if(m >= 0 && n >= 0 && m < width && n < height)
				{
					const float currentDepth = d_input[n*width+m];

					if (currentDepth != MINF) {
						const float weight = gaussD(sigmaD, m-x, n-y)*gaussR(sigmaR, currentDepth-depthCenter);

						sumWeight += weight;
						sum += weight*currentDepth;
					}
				}
			}
		}

		if(sumWeight > 0.0f) d_output[y*width+x] = sum / sumWeight;
	}
}

extern "C" void bilateralFilterFloatMap(float* d_output, float* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	bilateralFilterFloatMapDevice<<<gridSize, blockSize>>>(d_output, d_input, sigmaD, sigmaR, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Bilateral Filter Float4 Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void bilateralFilterFloat4MapDevice(float4* d_output, float4* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	//d_output[y*width+x] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	d_output[y*width+x] = make_float4(MINF, MINF, MINF, MINF);

	float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	float sumWeight = 0.0f;

	const float4 depthCenter = d_input[y*width+x];
	if (depthCenter.x != MINF) {
		for(int m = x-kernelRadius; m <= x+kernelRadius; m++)
		{
			for(int n = y-kernelRadius; n <= y+kernelRadius; n++)
			{		
				if(m >= 0 && n >= 0 && m < width && n < height)
				{
					const float4 currentDepth = d_input[n*width+m];

					if (currentDepth.x != MINF) {
						const float weight = gaussD(sigmaD, m-x, n-y)*gaussR(sigmaR, length(currentDepth-depthCenter));

						sum += weight*currentDepth;
						sumWeight += weight;
					}
				}
			}
		}
	}
	if(sumWeight > 0.0f) d_output[y*width+x] = sum / sumWeight;
}

extern "C" void bilateralFilterFloat4Map(float4* d_output, float4* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const dim3 gridSize(T_PER_BLOCK, T_PER_BLOCK);	
	const dim3 blockSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);	

	bilateralFilterFloat4MapDevice<<<gridSize, blockSize>>>(d_output, d_input, sigmaD, sigmaR, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//// Gauss Filter Float Map
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//__global__ void gaussFilterFloatMapDevice(float* d_output, float* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
//{
//	const int x = blockIdx.x*blockDim.x + threadIdx.x;
//	const int y = blockIdx.y*blockDim.y + threadIdx.y;
//
//	if(x >= width || y >= height) return;
//
//	const int kernelRadius = (int)ceil(2.0*sigmaD);
//
//	d_output[y*width+x] = MINF;
//
//	float sum = 0.0f;
//	float sumWeight = 0.0f;
//
//	const float depthCenter = d_input[y*width+x];
//	if(depthCenter != MINF)
//	{
//		for(int m = x-kernelRadius; m <= x+kernelRadius; m++)
//		{
//			for(int n = y-kernelRadius; n <= y+kernelRadius; n++)
//			{		
//				if(m >= 0 && n >= 0 && m < width && n < height)
//				{
//					const float currentDepth = d_input[n*width+m];
//
//					if(currentDepth != MINF && fabs(depthCenter-currentDepth) < sigmaR)
//					{
//						const float weight = gaussD(sigmaD, m-x, n-y);
//
//						sumWeight += weight;
//						sum += weight*currentDepth;
//					}
//				}
//			}
//		}
//	}
//
//	if(sumWeight > 0.0f) d_output[y*width+x] = sum / sumWeight;
//}
//
//extern "C" void gaussFilterFloatMap(float* d_output, float* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
//{
//	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
//	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);
//
//	gaussFilterFloatMapDevice<<<gridSize, blockSize>>>(d_output, d_input, sigmaD, sigmaR, width, height);
//	#ifdef _DEBUG
//		cutilSafeCall(hipDeviceSynchronize());
//		cutilCheckMsg(__FUNCTION__);
//	#endif
//}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Gauss Filter Float4 Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void gaussFilterFloat4MapDevice(float4* d_output, float4* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	//d_output[y*width+x] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	d_output[y*width+x] = make_float4(MINF, MINF, MINF, MINF);

	float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	float sumWeight = 0.0f;

	const float4 depthCenter = d_input[y*width+x];
	if (depthCenter.x != MINF) {
		for(int m = x-kernelRadius; m <= x+kernelRadius; m++)
		{
			for(int n = y-kernelRadius; n <= y+kernelRadius; n++)
			{		
				if(m >= 0 && n >= 0 && m < width && n < height)
				{
					const float4 currentDepth = d_input[n*width+m];

					if (currentDepth.x != MINF) {
						if(length(depthCenter-currentDepth) < sigmaR)
						{
							const float weight = gaussD(sigmaD, m-x, n-y);

							sumWeight += weight;
							sum += weight*currentDepth;
						}
					}
				}
			}
		}
	}

	if(sumWeight > 0.0f) d_output[y*width+x] = sum / sumWeight;
}

extern "C" void gaussFilterFloat4Map(float4* d_output, float4* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	gaussFilterFloat4MapDevice<<<gridSize, blockSize>>>(d_output, d_input, sigmaD, sigmaR, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Compute Normal Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void computeNormalsDevice(float4* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	d_output[y*width+x] = make_float4(MINF, MINF, MINF, MINF);

	if(x > 0 && x < width-1 && y > 0 && y < height-1)
	{
		const float4 CC = d_input[(y+0)*width+(x+0)];
		const float4 PC = d_input[(y+1)*width+(x+0)];
		const float4 CP = d_input[(y+0)*width+(x+1)];
		const float4 MC = d_input[(y-1)*width+(x+0)];
		const float4 CM = d_input[(y+0)*width+(x-1)];

		if(CC.x != MINF && PC.x != MINF && CP.x != MINF && MC.x != MINF && CM.x != MINF)
		{
			const float3 n = cross(make_float3(PC)-make_float3(MC), make_float3(CP)-make_float3(CM));
			const float  l = length(n);

			if(l > 0.0f)
			{
				d_output[y*width+x] = make_float4(n/-l, 1.0f);
			}
		}
	}
}

extern "C" void computeNormals(float4* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeNormalsDevice<<<gridSize, blockSize>>>(d_output, d_input, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Compute Normal Map 2
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void computeNormalsDevice2(float4* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	d_output[y*width+x] = make_float4(MINF, MINF, MINF, MINF);

	if(x > 0 && x < width-1 && y > 0 && y < height-1)
	{
		const float4 CC = d_input[(y+0)*width+(x+0)];
		const float4 MC = d_input[(y-1)*width+(x+0)];
		const float4 CM = d_input[(y+0)*width+(x-1)];

		if(CC.x != MINF && MC.x != MINF && CM.x != MINF)
		{
			const float3 n = cross(make_float3(MC)-make_float3(CC), make_float3(CM)-make_float3(CC));
			const float  l = length(n);

			if(l > 0.0f)
			{
				d_output[y*width+x] = make_float4(n/-l, 1.0f);
			}
		}
	}
}

extern "C" void computeNormals2(float4* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeNormalsDevice2<<<gridSize, blockSize>>>(d_output, d_input, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Compute Shading Value
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

inline __device__ void evaluateLightingModelTerms(float* d_out, float4 n)
{
	d_out[0] = 1.0;
	d_out[1] = n.y;
	d_out[2] = n.z;
	d_out[3] = n.x;
	d_out[4] = n.x*n.y;
	d_out[5] = n.y*n.z;
	d_out[6] = 3*n.z*n.z - 1;
	d_out[7] = n.z*n.x;
	d_out[8] = n.x*n.x-n.y*n.y;
}

inline __device__ float evaluateLightingModel(float* d_lit, float4 n)
{
	float tmp[9];
	evaluateLightingModelTerms(tmp, n);

	float sum = 0.0f;
	for(unsigned int i = 0; i<9; i++) sum += tmp[i]*d_lit[i];

	return sum;
}

__global__ void computeShadingValueDevice(float* d_outShading, float* d_indepth, float4* d_normals, float* d_clusterIDs, float* d_albedoEstimates, float4x4 Intrinsic, float* d_litcoeff, unsigned int width, unsigned int height)
{
	const unsigned int posx = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int posy = blockIdx.y*blockDim.y + threadIdx.y;

	if(posx >= width || posy >= height) return;

	d_outShading[posy*width+posx] = 0;

	if(posx > 0 && posx < width-1 && posy > 0 && posy < height-1)
	{
		float4 n = d_normals[posy*width+posx];

		if(n.x != MINF)
		{
			n.x = -n.x; // Change handedness
			n.z = -n.z;

			float albedo = d_albedoEstimates[(unsigned int)(d_clusterIDs[posy*width+posx]+0.5f)];
			float shadingval = albedo*evaluateLightingModel(d_litcoeff, n);

			if(shadingval<0.0f) shadingval = 0.0f;
			if(shadingval>1.0f) shadingval = 1.0f;

			d_outShading[posy*width+posx] = shadingval;
		}
	}
}

extern "C" void computeShadingValue(float* d_outShading, float* d_indepth, float4* d_normals, float* d_clusterIDs, float* d_albedoEstimates, float4x4 &Intrinsic, float* d_lighting, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeShadingValueDevice<<<gridSize, blockSize>>>(d_outShading, d_indepth, d_normals, d_clusterIDs, d_albedoEstimates, Intrinsic, d_lighting, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Simple Segmentation
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void computeSimpleSegmentationDevice(float* d_output, float* d_input, float depthThres, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	const float inputDepth = d_input[y*width+x];
	if(inputDepth != MINF && inputDepth < depthThres) d_output[y*width+x] = inputDepth;
	else											  d_output[y*width+x] = MINF;
}

extern "C" void computeSimpleSegmentation(float* d_output, float* d_input, float depthThres, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeSimpleSegmentationDevice<<<gridSize, blockSize>>>(d_output, d_input, depthThres, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Compute Edge Mask
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void computeMaskEdgeMapFloat4Device(unsigned char* d_output, float4* d_input, float* d_indepth, float threshold, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	d_output[y*width+x] = 1;
	d_output[width*height+y*width+x] = 1;

	const float thre = threshold *threshold *3.0f;
	if(x > 0 && y > 0 && x < width-1 && y < height-1)
	{	
		if(d_indepth[y*width+x] == MINF)
		{
			d_output[y*width+x] = 0;
			d_output[y*width+x-1] = 0;
			d_output[width*height+y*width+x] = 0;
			d_output[width*height+(y-1)*width+x] = 0;

			return;
		}

		const float4& p0 = d_input[(y+0)*width+(x+0)];
		const float4& p1 = d_input[(y+0)*width+(x+1)];
		const float4& p2 = d_input[(y+1)*width+(x+0)];

		float dU = sqrt(((p1.x-p0.x)*(p1.x-p0.x) + (p1.y-p0.y) * (p1.y-p0.y) + (p1.z-p0.z)*(p1.z-p0.z))/3.0f);
		float dV = sqrt(((p2.x-p0.x)*(p2.x-p0.x) + (p2.y-p0.y) * (p2.y-p0.y) + (p2.z-p0.z)*(p2.z-p0.z))/3.0f);

		//float dgradx = abs(d_indepth[y*width+x-1] + d_indepth[y*width+x+1] - 2.0f * d_indepth[y*width+x]);
		//float dgrady = abs(d_indepth[y*width+x-width] + d_indepth[y*width+x+width] - 2.0f * d_indepth[y*width+x]);


		if(dU > thre ) d_output[y*width+x] = 0;
		if(dV > thre ) d_output[width*height+y*width+x] = 0;

		//remove depth discontinuities
		const int r = 1;
		const float thres = 0.01f;

		const float pCC = d_indepth[y*width+x];
		for(int i = -r; i<=r; i++)
		{
			for(int j = -r; j<=r; j++)
			{
				int currentX = x+j;
				int currentY = y+i;

				if(currentX >= 0 && currentX < width && currentY >= 0 && currentY < height)
				{
					float d = d_indepth[currentY*width+currentX];

					if(d != MINF && abs(pCC-d) > thres)
					{
						d_output[y*width+x] = 0;
						d_output[width*height+y*width+x] = 0;
						return;
					}
				}
			}
		}
	}
}

extern "C" void computeMaskEdgeMapFloat4(unsigned char* d_output, float4* d_input, float* d_indepth, float threshold, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeMaskEdgeMapFloat4Device<<<gridSize, blockSize>>>(d_output, d_input, d_indepth, threshold,width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Clear Decission Array for Patch Depth Mask
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void clearDecissionArrayPatchDepthMaskDevice(int* d_output, unsigned int inputWidth, unsigned int inputHeight)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= 0 && x < inputWidth && y >= 0 && y < inputHeight) d_output[y*inputWidth+x] = 0;
}

extern "C" void clearDecissionArrayPatchDepthMask(int* d_output, unsigned int inputWidth, unsigned int inputHeight)
{
	const dim3 gridSize((inputWidth + T_PER_BLOCK - 1)/T_PER_BLOCK, (inputHeight + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	clearDecissionArrayPatchDepthMaskDevice<<<gridSize, blockSize>>>(d_output, inputWidth, inputHeight);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Decission Array for Patch Depth Mask
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void computeDecissionArrayPatchDepthMaskDevice(int* d_output, float* d_input, unsigned int patchSize, unsigned int inputWidth, unsigned int inputHeight)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= 0 && x < inputWidth && y >= 0 && y < inputHeight)
	{
		const int patchId_x = x/patchSize;
		const int patchId_y = y/patchSize;
		const int nPatchesWidth = (inputWidth+patchSize-1)/patchSize;

		const float d = d_input[y*inputWidth+x];
		if(d != MINF) atomicMax(&d_output[patchId_y*nPatchesWidth+patchId_x], 1);
	}
}

extern "C" void computeDecissionArrayPatchDepthMask(int* d_output, float* d_input, unsigned int patchSize, unsigned int inputWidth, unsigned int inputHeight)
{
	const dim3 gridSize((inputWidth + T_PER_BLOCK - 1)/T_PER_BLOCK, (inputHeight + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeDecissionArrayPatchDepthMaskDevice<<<gridSize, blockSize>>>(d_output, d_input, patchSize, inputWidth, inputHeight);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Compute Remapping Array for Patch Depth Mask
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void computeRemappingArrayPatchDepthMaskDevice(int* d_output, float* d_input, int* d_prefixSum, unsigned int patchSize, unsigned int inputWidth, unsigned int inputHeight)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= 0 && x < inputWidth && y >= 0 && y < inputHeight)
	{
		const int patchId_x = x/patchSize;
		const int patchId_y = y/patchSize;

		const int nPatchesWidth = (inputWidth+patchSize-1)/patchSize;

		const float d = d_input[y*inputWidth+x];
		if(d != MINF) d_output[d_prefixSum[patchId_y*nPatchesWidth+patchId_x]-1] = patchId_y*nPatchesWidth+patchId_x;
	}
}

extern "C" void computeRemappingArrayPatchDepthMask(int* d_output, float* d_input, int* d_prefixSum, unsigned int patchSize, unsigned int inputWidth, unsigned int inputHeight)
{
	const dim3 gridSize((inputWidth + T_PER_BLOCK - 1)/T_PER_BLOCK, (inputHeight + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeRemappingArrayPatchDepthMaskDevice<<<gridSize, blockSize>>>(d_output, d_input, d_prefixSum, patchSize, inputWidth, inputHeight);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Debug Patch Remap Array
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void DebugPatchRemapArrayDevice(float* d_mask, int* d_remapArray, unsigned int patchSize, unsigned int numElements, unsigned int inputWidth, unsigned int inputHeight)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;

	if(x < numElements)
	{
		int patchID = d_remapArray[x];

		const int nPatchesWidth = (inputWidth+patchSize-1)/patchSize;
		const int patchId_x = patchID%nPatchesWidth;
		const int patchId_y = patchID/nPatchesWidth;

		for(unsigned int i = 0; i<patchSize; i++)
		{
			for(unsigned int j = 0; j<patchSize; j++)
			{
				const int pixel_x = patchId_x*patchSize;
				const int pixel_y = patchId_y*patchSize;

				d_mask[(pixel_y+i)*inputWidth+(pixel_x+j)] = 3.0f;
			}
		}
	}
}

extern "C" void DebugPatchRemapArray(float* d_mask, int* d_remapArray, unsigned int patchSize, unsigned int numElements, unsigned int inputWidth, unsigned int inputHeight)
{
	const dim3 gridSize((numElements + T_PER_BLOCK*T_PER_BLOCK - 1)/(T_PER_BLOCK*T_PER_BLOCK));
	const dim3 blockSize(T_PER_BLOCK*T_PER_BLOCK);

	DebugPatchRemapArrayDevice<<<gridSize, blockSize>>>(d_mask, d_remapArray, patchSize, numElements, inputWidth, inputHeight);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Resample Float Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

inline __device__ float bilinearInterpolationFloat(float x, float y, float* d_input, unsigned int imageWidth, unsigned int imageHeight)
{
	const int2 p00 = make_int2(floor(x), floor(y));
	const int2 p01 = p00 + make_int2(0.0f, 1.0f);
	const int2 p10 = p00 + make_int2(1.0f, 0.0f);
	const int2 p11 = p00 + make_int2(1.0f, 1.0f);

	const float alpha = x - p00.x;
	const float beta  = y - p00.y;

	float s0 = 0.0f; float w0 = 0.0f;
	if(p00.x < imageWidth && p00.y < imageHeight) { float v00 = d_input[p00.y*imageWidth + p00.x]; if(v00 != MINF) { s0 += (1.0f-alpha)*v00; w0 += (1.0f-alpha); } }
	if(p10.x < imageWidth && p10.y < imageHeight) { float v10 = d_input[p10.y*imageWidth + p10.x]; if(v10 != MINF) { s0 +=		 alpha *v10; w0 +=		 alpha ; } }

	float s1 = 0.0f; float w1 = 0.0f;
	if(p01.x < imageWidth && p01.y < imageHeight) { float v01 = d_input[p01.y*imageWidth + p01.x]; if(v01 != MINF) { s1 += (1.0f-alpha)*v01; w1 += (1.0f-alpha);} }
	if(p11.x < imageWidth && p11.y < imageHeight) { float v11 = d_input[p11.y*imageWidth + p11.x]; if(v11 != MINF) { s1 +=		 alpha *v11; w1 +=		 alpha ;} }

	const float p0 = s0/w0;
	const float p1 = s1/w1;

	float ss = 0.0f; float ww = 0.0f;
	if(w0 > 0.0f) { ss += (1.0f-beta)*p0; ww += (1.0f-beta); }
	if(w1 > 0.0f) { ss +=		beta *p1; ww +=		  beta ; }

	if(ww > 0.0f) return ss/ww;
	else		  return MINF;
}

__global__ void resampleFloatMapDevice(float* d_colorMapResampledFloat, float* d_colorMapFloat, unsigned int inputWidth, unsigned int inputHeight, unsigned int outputWidth, unsigned int outputHeight)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x < outputWidth && y < outputHeight)
	{
		const float scaleWidth  = (float)(inputWidth-1) /(float)(outputWidth-1);
		const float scaleHeight = (float)(inputHeight-1)/(float)(outputHeight-1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth +0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight+0.5f);

		if(xInput < inputWidth && yInput < inputHeight)
		{
			d_colorMapResampledFloat[y*outputWidth+x] = bilinearInterpolationFloat(x*scaleWidth, y*scaleHeight, d_colorMapFloat, inputWidth, inputHeight);
		}
	}
}

extern "C" void resampleFloatMap(float* d_colorMapResampledFloat, unsigned int outputWidth, unsigned int outputHeight, float* d_colorMapFloat, unsigned int inputWidth, unsigned int inputHeight)
{
	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1)/T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleFloatMapDevice<<<gridSize, blockSize>>>(d_colorMapResampledFloat, d_colorMapFloat, inputWidth, inputHeight, outputWidth, outputHeight);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Resample Float4 Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

inline __device__ float4 bilinearInterpolationFloat4(float x, float y, float4* d_input, unsigned int imageWidth, unsigned int imageHeight)
{
	const int2 p00 = make_int2(floor(x), floor(y));
	const int2 p01 = p00 + make_int2(0.0f, 1.0f);
	const int2 p10 = p00 + make_int2(1.0f, 0.0f);
	const int2 p11 = p00 + make_int2(1.0f, 1.0f);

	const float alpha = x - p00.x;
	const float beta  = y - p00.y;

	//const float INVALID = 0.0f;
	const float INVALID = MINF;

	float4 s0 = make_float4(0.0f, 0.0f, 0.0f, 0.0f); float w0 = 0.0f;
	if(p00.x < imageWidth && p00.y < imageHeight) { float4 v00 = d_input[p00.y*imageWidth + p00.x]; if(v00.x != INVALID && v00.y != INVALID && v00.z != INVALID) { s0 += (1.0f-alpha)*v00; w0 += (1.0f-alpha); } }
	if(p10.x < imageWidth && p10.y < imageHeight) { float4 v10 = d_input[p10.y*imageWidth + p10.x]; if(v10.x != INVALID && v10.y != INVALID && v10.z != INVALID) { s0 +=		alpha *v10; w0 +=		alpha ; } }

	float4 s1 = make_float4(0.0f, 0.0f, 0.0f, 0.0f); float w1 = 0.0f;
	if(p01.x < imageWidth && p01.y < imageHeight) { float4 v01 = d_input[p01.y*imageWidth + p01.x]; if(v01.x != INVALID && v01.y != INVALID && v01.z != INVALID) { s1 += (1.0f-alpha)*v01; w1 += (1.0f-alpha);} }
	if(p11.x < imageWidth && p11.y < imageHeight) { float4 v11 = d_input[p11.y*imageWidth + p11.x]; if(v11.x != INVALID && v11.y != INVALID && v11.z != INVALID) { s1 +=		alpha *v11; w1 +=		alpha ;} }

	const float4 p0 = s0/w0;
	const float4 p1 = s1/w1;

	float4 ss = make_float4(0.0f, 0.0f, 0.0f, 0.0f); float ww = 0.0f;
	if(w0 > 0.0f) { ss += (1.0f-beta)*p0; ww += (1.0f-beta); }
	if(w1 > 0.0f) { ss +=		beta *p1; ww +=		  beta ; }

	if(ww > 0.0f) return ss/ww;
	else		  return make_float4(MINF, MINF, MINF, MINF);
}

__global__ void resampleFloat4MapDevice(float4* d_colorMapResampledFloat4, float4* d_colorMapFloat4, unsigned int inputWidth, unsigned int inputHeight, unsigned int outputWidth, unsigned int outputHeight)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x < outputWidth && y < outputHeight)
	{
		const float scaleWidth  = (float)(inputWidth-1) /(float)(outputWidth-1);
		const float scaleHeight = (float)(inputHeight-1)/(float)(outputHeight-1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth +0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight+0.5f);

		if(xInput < inputWidth && yInput < inputHeight)
		{
			d_colorMapResampledFloat4[y*outputWidth+x] = bilinearInterpolationFloat4(x*scaleWidth, y*scaleHeight, d_colorMapFloat4, inputWidth, inputHeight);
		}
	}
}

extern "C" void resampleFloat4Map(float4* d_colorMapResampledFloat4, unsigned int outputWidth, unsigned int outputHeight, float4* d_colorMapFloat4, unsigned int inputWidth, unsigned int inputHeight)
{
	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1)/T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleFloat4MapDevice<<<gridSize, blockSize>>>(d_colorMapResampledFloat4, d_colorMapFloat4, inputWidth, inputHeight, outputWidth, outputHeight);
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Resample Unsigned Char Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void downsampleUnsignedCharMapDevice(unsigned char* d_MapResampled, unsigned char* d_Map, unsigned int inputWidth, unsigned int inputHeight, unsigned int outputWidth, unsigned int outputHeight, unsigned int layerOffsetInput, unsigned int layerOffsetOutput)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= outputWidth || y >= outputHeight) return;

	unsigned char res = 0;

	const unsigned int inputX = 2*x;
	const unsigned int inputY = 2*y;

	if((inputY+0) < inputHeight && (inputX+0) < inputWidth)	res += d_Map[layerOffsetInput + ((inputY+0)*inputWidth + (inputX+0))];
	if((inputY+0) < inputHeight && (inputX+1) < inputWidth)	res += d_Map[layerOffsetInput + ((inputY+0)*inputWidth + (inputX+1))];
	if((inputY+1) < inputHeight && (inputX+0) < inputWidth)	res += d_Map[layerOffsetInput + ((inputY+1)*inputWidth + (inputX+0))];
	if((inputY+1) < inputHeight && (inputX+1) < inputWidth) res += d_Map[layerOffsetInput + ((inputY+1)*inputWidth + (inputX+1))];

	if(res == 4) d_MapResampled[layerOffsetOutput+(y*outputWidth+x)] = 1;
	else		 d_MapResampled[layerOffsetOutput+(y*outputWidth+x)] = 0;
}

extern "C" void downsampleUnsignedCharMap(unsigned char* d_MapResampled, unsigned int outputWidth, unsigned int outputHeight, unsigned char* d_Map, unsigned int inputWidth, unsigned int inputHeight)
{
	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1)/T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	downsampleUnsignedCharMapDevice<<<gridSize, blockSize>>>(d_MapResampled, d_Map, inputWidth, inputHeight, outputWidth, outputHeight, 0, 0);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

	downsampleUnsignedCharMapDevice<<<gridSize, blockSize>>>(d_MapResampled, d_Map, inputWidth, inputHeight, outputWidth, outputHeight, inputWidth*inputHeight, outputWidth*outputHeight);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Convert Edge Mask to Float Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void convertEdgeMaskToFloatDevice(float* d_output, unsigned char* d_input, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	d_output[y*width+x] = min(d_input[y*width+x], d_input[width*height+y*width+x]);
}

extern "C" void convertEdgeMaskToFloat(float* d_output, unsigned char* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertEdgeMaskToFloatDevice<<<gridSize, blockSize>>>(d_output, d_input, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Dilate Depth Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void dilateDepthMapDevice(float* d_output, float* d_input, float* d_inputOrig, int structureSize, int width, int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= 0 && x < width && y >= 0 && y < height)
	{
		float sum = 0.0f;
		float count = 0.0f;
		float oldDepth = d_inputOrig[y*width+x];
		if(oldDepth != MINF && oldDepth != 0)
		{
			for(int i = -structureSize; i<=structureSize; i++)
			{
				for(int j = -structureSize; j<=structureSize; j++)
				{
					if(x+j >= 0 && x+j < width && y+i >= 0 && y+i < height)
					{
						const float d = d_input[(y+i)*width+(x+j)];

						if(d != MINF && d != 0.0f && fabs(d-oldDepth) < 0.05f)
						{
							sum += d;
							count += 1.0f;
						}
					}
				}
			}
		}

		if(count > ((2*structureSize+1)*(2*structureSize+1))/36) d_output[y*width+x] = 1.0f;
		else			 d_output[y*width+x] = MINF;
	}
}

extern "C" void dilateDepthMapMask(float* d_output, float* d_input, float* d_inputOrig, int structureSize, int width, int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	dilateDepthMapDevice<<<gridSize, blockSize>>>(d_output, d_input, d_inputOrig, structureSize, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Mean Filter Depth Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void removeDevMeanMapMaskDevice(float* d_output, float* d_input, int structureSize, int width, int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	d_output[y*width+x] = d_input[y*width+x];

	if(x >= 0 && x < width && y >= 0 && y < height)
	{
		float oldDepth = d_input[y*width+x];

		float mean = 0.0f;
		float meanSquared = 0.0f;
		float count = 0.0f;
		for(int i = -structureSize; i<=structureSize; i++)
		{
			for(int j = -structureSize; j<=structureSize; j++)
			{
				if(x+j >= 0 && x+j < width && y+i >= 0 && y+i < height)
				{
					float depth = d_input[(y+i)*width+(x+j)];
					if(depth == MINF)
					{
						depth = 8.0f;
					}

					if(depth > 0.0f)
					{
						mean		+= depth;
						meanSquared += depth*depth;
						count		+= 1.0f;
					}
				}
			}
		}

		mean/=count;
		meanSquared/=count;

		float stdDev = sqrt(meanSquared-mean*mean);

		if(fabs(oldDepth-mean) > 0.5f*stdDev)// || stdDev> 0.005f)
		{
			d_output[y*width+x] = MINF;
		}
	}
}

extern "C" void removeDevMeanMapMask(float* d_output, float* d_input, int structureSize, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	removeDevMeanMapMaskDevice<<<gridSize, blockSize>>>(d_output, d_input, structureSize, width, height);
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}




// Nearest neighbour
inline __device__ bool getValueNearestNeighbourNoCheck(const float2& p, const float4* inputMap, unsigned int imageWidth, unsigned int imageHeight, float4* outValue)
{
	const int u = (int)(p.x + 0.5f);
	const int v = (int)(p.y + 0.5f);

	if(u < 0 || u > imageWidth || v < 0 || v > imageHeight) return false;

	*outValue = inputMap[v*imageWidth + u];

	return true;
}

inline __device__ bool getValueNearestNeighbour(const float2& p, const float4* inputMap, unsigned int imageWidth, unsigned int imageHeight, float4* outValue)
{
	bool valid = getValueNearestNeighbourNoCheck(p, inputMap, imageWidth, imageHeight, outValue);
	return valid && (outValue->x != MINF && outValue->y != MINF && outValue->z != MINF);
}

// Nearest neighbour
inline __device__ bool getValueNearestNeighbourFloatNoCheck(const float2& p, const float* inputMap, unsigned int imageWidth, unsigned int imageHeight, float* outValue)
{
	const int u = (int)(p.x + 0.5f);
	const int v = (int)(p.y + 0.5f);

	if(u < 0 || u > imageWidth || v < 0 || v > imageHeight) return false;

	*outValue = inputMap[v*imageWidth + u];

	return true;
}

inline __device__ bool getValueNearestNeighbourFloat(const float2& p, const float* inputMap, unsigned int imageWidth, unsigned int imageHeight, float* outValue)
{
	bool valid = getValueNearestNeighbourFloatNoCheck(p, inputMap, imageWidth, imageHeight, outValue);
	return valid && (*outValue != MINF);
}

/////////////////////////////////////////////
// Compute derivatives in camera space
/////////////////////////////////////////////

__global__ void computeDerivativesCameraSpaceDevice(float4* d_positions, unsigned int imageWidth, unsigned int imageHeight, float4* d_positionsDU, float4* d_positionsDV)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	const int index = y*imageWidth+x;

	if(x >= 0 && x < imageWidth && y >= 0 && y < imageHeight)
	{
		d_positionsDU[index] = make_float4(MINF, MINF, MINF, MINF);
		d_positionsDV[index] = make_float4(MINF, MINF, MINF, MINF);
	
		if(x > 0 && x < imageWidth - 1 && y > 0 && y < imageHeight - 1)
		{
			float4 pos00; bool valid00 = getValueNearestNeighbour(make_float2(x-1, y-1), d_positions, imageWidth, imageHeight, &pos00); if(!valid00) return;
			float4 pos01; bool valid01 = getValueNearestNeighbour(make_float2(x-1, y-0), d_positions, imageWidth, imageHeight, &pos01); if(!valid01) return;
			float4 pos02; bool valid02 = getValueNearestNeighbour(make_float2(x-1, y+1), d_positions, imageWidth, imageHeight, &pos02); if(!valid02) return;
			
			float4 pos10; bool valid10 = getValueNearestNeighbour(make_float2(x-0, y-1), d_positions, imageWidth, imageHeight, &pos10); if(!valid10) return;
			float4 pos11; bool valid11 = getValueNearestNeighbour(make_float2(x-0, y-0), d_positions, imageWidth, imageHeight, &pos11); if(!valid11) return;
			float4 pos12; bool valid12 = getValueNearestNeighbour(make_float2(x-0, y+1), d_positions, imageWidth, imageHeight, &pos12); if(!valid12) return;

			float4 pos20; bool valid20 = getValueNearestNeighbour(make_float2(x+1, y-1), d_positions, imageWidth, imageHeight, &pos20); if(!valid20) return;
			float4 pos21; bool valid21 = getValueNearestNeighbour(make_float2(x+1, y-0), d_positions, imageWidth, imageHeight, &pos21); if(!valid21) return;
			float4 pos22; bool valid22 = getValueNearestNeighbour(make_float2(x+1, y+1), d_positions, imageWidth, imageHeight, &pos22); if(!valid22) return;

			float4 resU = (-1.0f)*pos00 + (1.0f)*pos20 +
						  (-2.0f)*pos01 + (2.0f)*pos21 +
						  (-1.0f)*pos02 + (1.0f)*pos22;
			resU /= 8.0f;
			
			float4 resV = (-1.0f)*pos00 + (-2.0f)*pos10 + (-1.0f)*pos20 + 
						  ( 1.0f)*pos02 + ( 2.0f)*pos12 + ( 1.0f)*pos22;
			resV /= 8.0f;

			//if(mat3x1(make_float3(resU)).norm1D() > 0.02f) return;
			//if(mat3x1(make_float3(resV)).norm1D() > 0.02f) return;

			d_positionsDU[index] = resU;
			d_positionsDV[index] = resV;
		}
	}
}

extern "C" void computeDerivativesCameraSpace(float4* d_positions, unsigned int imageWidth, unsigned int imageHeight, float4* d_positionsDU, float4* d_positionsDV)
{

	const dim3 gridSize((imageWidth + T_PER_BLOCK - 1)/T_PER_BLOCK, (imageHeight + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeDerivativesCameraSpaceDevice<<<gridSize, blockSize>>>(d_positions, imageWidth, imageHeight, d_positionsDU, d_positionsDV);
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}


/////////////////////////////////////////////
// Compute Intensity and Derivatives
/////////////////////////////////////////////

__global__ void computeIntensityAndDerivativesDevice(float* d_intensity, unsigned int imageWidth, unsigned int imageHeight, float4* d_intensityAndDerivatives)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	const int index = y*imageWidth+x;

	if(x >= 0 && x < imageWidth && y >= 0 && y < imageHeight)
	{
		d_intensityAndDerivatives[index] = make_float4(MINF, MINF, MINF, MINF);
			
		if(x > 0 && x < imageWidth - 1 && y > 0 && y < imageHeight - 1)
		{
			float pos00; bool valid00 = getValueNearestNeighbourFloat(make_float2(x-1, y-1), d_intensity, imageWidth, imageHeight, &pos00); if(!valid00) return;
			float pos01; bool valid01 = getValueNearestNeighbourFloat(make_float2(x-1, y-0), d_intensity, imageWidth, imageHeight, &pos01); if(!valid01) return;
			float pos02; bool valid02 = getValueNearestNeighbourFloat(make_float2(x-1, y+1), d_intensity, imageWidth, imageHeight, &pos02); if(!valid02) return;
			
			float pos10; bool valid10 = getValueNearestNeighbourFloat(make_float2(x-0, y-1), d_intensity, imageWidth, imageHeight, &pos10); if(!valid10) return;
			float pos11; bool valid11 = getValueNearestNeighbourFloat(make_float2(x-0, y-0), d_intensity, imageWidth, imageHeight, &pos11); if(!valid11) return;
			float pos12; bool valid12 = getValueNearestNeighbourFloat(make_float2(x-0, y+1), d_intensity, imageWidth, imageHeight, &pos12); if(!valid12) return;

			float pos20; bool valid20 = getValueNearestNeighbourFloat(make_float2(x+1, y-1), d_intensity, imageWidth, imageHeight, &pos20); if(!valid20) return;
			float pos21; bool valid21 = getValueNearestNeighbourFloat(make_float2(x+1, y-0), d_intensity, imageWidth, imageHeight, &pos21); if(!valid21) return;
			float pos22; bool valid22 = getValueNearestNeighbourFloat(make_float2(x+1, y+1), d_intensity, imageWidth, imageHeight, &pos22); if(!valid22) return;

			float resU = (-1.0f)*pos00 + (1.0f)*pos20 +
						  (-2.0f)*pos01 + (2.0f)*pos21 +
						  (-1.0f)*pos02 + (1.0f)*pos22;
			resU /= 8.0f;
			
			float resV = (-1.0f)*pos00 + (-2.0f)*pos10 + (-1.0f)*pos20 + 
						  ( 1.0f)*pos02 + ( 2.0f)*pos12 + ( 1.0f)*pos22;
			resV /= 8.0f;

			d_intensityAndDerivatives[index] = make_float4(pos11, resU, resV, 1.0f);
		}
	}
}

extern "C" void computeIntensityAndDerivatives(float* d_intensity, unsigned int imageWidth, unsigned int imageHeight, float4* d_intensityAndDerivatives)
{
	const dim3 gridSize((imageWidth + T_PER_BLOCK - 1)/T_PER_BLOCK, (imageHeight + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeIntensityAndDerivativesDevice<<<gridSize, blockSize>>>(d_intensity, imageWidth, imageHeight, d_intensityAndDerivatives);
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}


/////////////////////////////////////////////
// Compute grdient intensity magnitude
/////////////////////////////////////////////

__global__ void computeGradientIntensityMagnitudeDevice(float4* d_inputDU, float4* d_inputDV, unsigned int imageWidth, unsigned int imageHeight, float4* d_ouput)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	const int index = y*imageWidth+x;

	d_ouput[index] = make_float4(MINF, MINF, MINF, MINF);

	float4 DU = d_inputDU[index];
	float4 DV = d_inputDV[index];

	if(DU.x != MINF && DV.x != MINF)
	{
		float m = sqrtf(DU.x*DU.x+DV.x*DV.x);

		if(m > 0.005f)
		{
			d_ouput[index] = make_float4(m, m, m, 1.0f);
		}
	}
}

extern "C" void computeGradientIntensityMagnitude(float4* d_inputDU, float4* d_inputDV, unsigned int imageWidth, unsigned int imageHeight, float4* d_ouput)
{
	const dim3 gridSize((imageWidth + T_PER_BLOCK - 1)/T_PER_BLOCK, (imageHeight + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeGradientIntensityMagnitudeDevice<<<gridSize, blockSize>>>(d_inputDU, d_inputDV, imageWidth, imageHeight, d_ouput);
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

/////////////////////////////////////////////
// Transform
/////////////////////////////////////////////

__global__ void transformCameraSpaceMapDevice(float4* d_positions, unsigned int imageWidth, unsigned int imageHeight,  float4* d_output)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	const int index = y*imageWidth+x;

	if(x >= 0 && x < imageWidth && y >= 0 && y < imageHeight)
	{
		d_output[index] = d_positions[index];

		if(d_positions[index].x != MINF && d_positions[index].y != MINF && d_positions[index].z != MINF)
		{
			d_output[index] =  d_positions[index]+make_float4(0.0f, 0.0f, 0.0f, 0.0f);
		}		
	}
}

extern "C" void transformCameraSpaceMap(float4* d_positions, unsigned int imageWidth, unsigned int imageHeight,  float4* d_output)
{
	const dim3 gridSize((imageWidth + T_PER_BLOCK - 1)/T_PER_BLOCK, (imageHeight + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	transformCameraSpaceMapDevice<<<gridSize, blockSize>>>(d_positions, imageWidth, imageHeight, d_output);
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

















////////////////////////////////////
// Depth to HSV map conversion /////
////////////////////////////////////

__device__ float3 convertHSVToRGB(const float3& hsv) {
	float H = hsv.x;
	float S = hsv.y;
	float V = hsv.z;

	float hd = H/60.0f;
	unsigned int h = (unsigned int)hd;
	float f = hd-h;

	float p = V*(1.0f-S);
	float q = V*(1.0f-S*f);
	float t = V*(1.0f-S*(1.0f-f));

	if(h == 0 || h == 6)
	{
		return make_float3(V, t, p);
	}
	else if(h == 1)
	{
		return make_float3(q, V, p);
	}
	else if(h == 2)
	{
		return make_float3(p, V, t);
	}
	else if(h == 3)
	{
		return make_float3(p, q, V);
	}
	else if(h == 4)
	{
		return make_float3(t, p, V);
	}
	else
	{
		return make_float3(V, p, q);
	}
}


__device__ float3 convertDepthToRGB(float depth, float depthMin, float depthMax) {
	float depthZeroOne = (depth - depthMin)/(depthMax - depthMin);
	float x = 1.0f-depthZeroOne;
	if (x < 0.0f)	x = 0.0f;
	if (x > 1.0f)	x = 1.0f;
	//return convertHSVToRGB(make_float3(240.0f*x, 1.0f, 0.5f));
	x = 360.0f*x - 120.0f;
	if (x < 0.0f) x += 359.0f;
	return convertHSVToRGB(make_float3(x, 1.0f, 0.5f));
}

__global__ void depthToHSVDevice(float4* d_output, const float* d_input, unsigned int width, unsigned int height, float minDepth, float maxDepth)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;
	
	if (x >= 0 && x < width && y >= 0 && y < height) {
		
		float depth = d_input[y*width + x];
		if (depth != MINF && depth != 0.0f && depth >= minDepth && depth <= maxDepth) {
			float3 c = convertDepthToRGB(depth, minDepth, maxDepth);
			d_output[y*width + x] = make_float4(c, 1.0f);
		} else {
			d_output[y*width + x] = make_float4(0.0f);
		}
	}
}

extern "C" void depthToHSV(float4* d_output, const float* d_input, unsigned int width, unsigned int height, float minDepth, float maxDepth) {
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	depthToHSVDevice<<<gridSize, blockSize>>>(d_output, d_input, width, height, minDepth, maxDepth);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}





#endif // _CAMERA_UTIL_
