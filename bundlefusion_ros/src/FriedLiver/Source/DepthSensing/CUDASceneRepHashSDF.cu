#include "hip/hip_runtime.h"

#include <cutil_inline.h>
#include <cutil_math.h>

#include "cuda_SimpleMatrixUtil.h"

#include "VoxelUtilHashSDF.h"
#include "DepthCameraUtil.h"

#define T_PER_BLOCK 8

texture<float, hipTextureType2D, hipReadModeElementType> depthTextureRef;
texture<uchar4, hipTextureType2D, hipReadModeElementType> colorTextureRef;

extern "C" void bindInputDepthColorTextures(const DepthCameraData& depthCameraData, unsigned int width, unsigned int height) 
{
	//cutilSafeCall(hipBindTextureToArray(depthTextureRef, depthCameraData.d_depthArray, depthCameraData.h_depthChannelDesc));
	//cutilSafeCall(hipBindTextureToArray(colorTextureRef, depthCameraData.d_colorArray, depthCameraData.h_colorChannelDesc));

	cutilSafeCall(hipBindTexture2D(0, depthTextureRef, depthCameraData.d_depthData, depthTextureRef.channelDesc, width, height, sizeof(float)*width));
	cutilSafeCall(hipBindTexture2D(0, colorTextureRef, depthCameraData.d_colorData, colorTextureRef.channelDesc, width, height, sizeof(uchar4)*width));

	depthTextureRef.filterMode = hipFilterModePoint;
	colorTextureRef.filterMode = hipFilterModePoint;
}

__global__ void resetHeapKernel(HashDataStruct hashData) 
{
	const HashParams& hashParams = c_hashParams;
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx == 0) {
		hashData.d_heapCounter[0] = hashParams.m_numSDFBlocks - 1;	//points to the last element of the array
	}
	
	if (idx < hashParams.m_numSDFBlocks) {

		hashData.d_heap[idx] = hashParams.m_numSDFBlocks - idx - 1;
		uint blockSize = SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;
		uint base_idx = idx * blockSize;
		for (uint i = 0; i < blockSize; i++) {
			hashData.deleteVoxel(base_idx+i);
		}
	}
}

__global__ void resetHashKernel(HashDataStruct hashData) 
{
	const HashParams& hashParams = c_hashParams;
	const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < hashParams.m_hashNumBuckets * HASH_BUCKET_SIZE) {
		hashData.deleteHashEntry(hashData.d_hash[idx]);
		hashData.deleteHashEntry(hashData.d_hashCompactified[idx]);
	}
}


__global__ void resetHashBucketMutexKernel(HashDataStruct hashData) 
{
	const HashParams& hashParams = c_hashParams;
	const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < hashParams.m_hashNumBuckets) {
		hashData.d_hashBucketMutex[idx] = FREE_ENTRY;
	}
}

extern "C" void resetCUDA(HashDataStruct& hashData, const HashParams& hashParams)
{
	{
		//resetting the heap and SDF blocks
		const dim3 gridSize((hashParams.m_numSDFBlocks + (T_PER_BLOCK*T_PER_BLOCK) - 1)/(T_PER_BLOCK*T_PER_BLOCK), 1);
		const dim3 blockSize((T_PER_BLOCK*T_PER_BLOCK), 1);

		resetHeapKernel<<<gridSize, blockSize>>>(hashData);


		#ifdef _DEBUG
			cutilSafeCall(hipDeviceSynchronize());
			cutilCheckMsg(__FUNCTION__);
		#endif

	}

	{
		//resetting the hash
		const dim3 gridSize((HASH_BUCKET_SIZE * hashParams.m_hashNumBuckets + (T_PER_BLOCK*T_PER_BLOCK) - 1)/(T_PER_BLOCK*T_PER_BLOCK), 1);
		const dim3 blockSize((T_PER_BLOCK*T_PER_BLOCK), 1);

		resetHashKernel<<<gridSize, blockSize>>>(hashData);

		#ifdef _DEBUG
			cutilSafeCall(hipDeviceSynchronize());
			cutilCheckMsg(__FUNCTION__);
		#endif
	}

	{
		//resetting the mutex
		const dim3 gridSize((hashParams.m_hashNumBuckets + (T_PER_BLOCK*T_PER_BLOCK) - 1)/(T_PER_BLOCK*T_PER_BLOCK), 1);
		const dim3 blockSize((T_PER_BLOCK*T_PER_BLOCK), 1);

		resetHashBucketMutexKernel<<<gridSize, blockSize>>>(hashData);

		#ifdef _DEBUG
			cutilSafeCall(hipDeviceSynchronize());
			cutilCheckMsg(__FUNCTION__);
		#endif
	}


}

extern "C" void resetHashBucketMutexCUDA(HashDataStruct& hashData, const HashParams& hashParams)
{
	const dim3 gridSize((hashParams.m_hashNumBuckets + (T_PER_BLOCK*T_PER_BLOCK) - 1)/(T_PER_BLOCK*T_PER_BLOCK), 1);
	const dim3 blockSize((T_PER_BLOCK*T_PER_BLOCK), 1);

	resetHashBucketMutexKernel<<<gridSize, blockSize>>>(hashData);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}


__device__
unsigned int linearizeChunkPos(const int3& chunkPos)
{
	int3 p = chunkPos-c_hashParams.m_streamingMinGridPos;
	return  p.z * c_hashParams.m_streamingGridDimensions.x * c_hashParams.m_streamingGridDimensions.y +
			p.y * c_hashParams.m_streamingGridDimensions.x +
			p.x;
}

__device__
int3 worldToChunks(const float3& posWorld)
{
	float3 p;
	p.x = posWorld.x/c_hashParams.m_streamingVoxelExtents.x;
	p.y = posWorld.y/c_hashParams.m_streamingVoxelExtents.y;
	p.z = posWorld.z/c_hashParams.m_streamingVoxelExtents.z;

	float3 s;
	s.x = (float)sign(p.x);
	s.y = (float)sign(p.y);
	s.z = (float)sign(p.z);

	return make_int3(p+s*0.5f);
}

__device__
bool isSDFBlockStreamedOut(const int3& sdfBlock, const HashDataStruct& hashData, const unsigned int* d_bitMask)	//TODO MATTHIAS (-> move to HashData)
{
	if (!d_bitMask) return false;	//TODO can statically disable streaming??


	float3 posWorld = hashData.virtualVoxelPosToWorld(hashData.SDFBlockToVirtualVoxelPos(sdfBlock)); // sdfBlock is assigned to chunk by the bottom right sample pos

	uint index = linearizeChunkPos(worldToChunks(posWorld));
	uint nBitsInT = 32;
	return ((d_bitMask[index/nBitsInT] & (0x1 << (index%nBitsInT))) != 0x0);
}

__global__ void allocKernel(HashDataStruct hashData, DepthCameraData cameraData, const unsigned int* d_bitMask) 
{
	const HashParams& hashParams = c_hashParams;
	const DepthCameraParams& cameraParams = c_depthCameraParams;

	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	
	if (x < cameraParams.m_imageWidth && y < cameraParams.m_imageHeight)
	{

		float d = tex2D(depthTextureRef, x, y);
		
		//if (d == MINF || d < cameraParams.m_sensorDepthWorldMin || d > cameraParams.m_sensorDepthWorldMax)	return;
		if (d == MINF || d == 0.0f)	return;


		if (d >= hashParams.m_maxIntegrationDistance) return;

		float t = hashData.getTruncation(d);
		float minDepth = min(hashParams.m_maxIntegrationDistance, d-t);
		float maxDepth = min(hashParams.m_maxIntegrationDistance, d+t);
		if (minDepth >= maxDepth) return;

		float3 rayMin = DepthCameraData::kinectDepthToSkeleton(x, y, minDepth);
		rayMin = hashParams.m_rigidTransform * rayMin;
		float3 rayMax = DepthCameraData::kinectDepthToSkeleton(x, y, maxDepth);
		rayMax = hashParams.m_rigidTransform * rayMax;

		
		float3 rayDir = normalize(rayMax - rayMin);
	
		int3 idCurrentVoxel = hashData.worldToSDFBlock(rayMin);
		int3 idEnd = hashData.worldToSDFBlock(rayMax);
		
		float3 step = make_float3(sign(rayDir));
		float3 boundaryPos = hashData.SDFBlockToWorld(idCurrentVoxel+make_int3(clamp(step, 0.0, 1.0f)))-0.5f*hashParams.m_virtualVoxelSize;
		float3 tMax = (boundaryPos-rayMin)/rayDir;
		float3 tDelta = (step*SDF_BLOCK_SIZE*hashParams.m_virtualVoxelSize)/rayDir;
		int3 idBound = make_int3(make_float3(idEnd)+step);

		//#pragma unroll
		//for(int c = 0; c < 3; c++) {
		//	if (rayDir[c] == 0.0f) { tMax[c] = PINF; tDelta[c] = PINF; }
		//	if (boundaryPos[c] - rayMin[c] == 0.0f) { tMax[c] = PINF; tDelta[c] = PINF; }
		//}
		if (rayDir.x == 0.0f) { tMax.x = PINF; tDelta.x = PINF; }
		if (boundaryPos.x - rayMin.x == 0.0f) { tMax.x = PINF; tDelta.x = PINF; }

		if (rayDir.y == 0.0f) { tMax.y = PINF; tDelta.y = PINF; }
		if (boundaryPos.y - rayMin.y == 0.0f) { tMax.y = PINF; tDelta.y = PINF; }

		if (rayDir.z == 0.0f) { tMax.z = PINF; tDelta.z = PINF; }
		if (boundaryPos.z - rayMin.z == 0.0f) { tMax.z = PINF; tDelta.z = PINF; }


		unsigned int iter = 0; // iter < g_MaxLoopIterCount
		unsigned int g_MaxLoopIterCount = 1024;	//TODO MATTHIAS MOVE TO GLOBAL APP STATE
#pragma unroll 1
		while(iter < g_MaxLoopIterCount) {

			//check if it's in the frustum and not checked out
			if (hashData.isSDFBlockInCameraFrustumApprox(idCurrentVoxel) && !isSDFBlockStreamedOut(idCurrentVoxel, hashData, d_bitMask)) {		
				hashData.allocBlock(idCurrentVoxel);
			}

			// Traverse voxel grid
			if(tMax.x < tMax.y && tMax.x < tMax.z)	{
				idCurrentVoxel.x += step.x;
				if(idCurrentVoxel.x == idBound.x) return;
				tMax.x += tDelta.x;
			}
			else if(tMax.z < tMax.y) {
				idCurrentVoxel.z += step.z;
				if(idCurrentVoxel.z == idBound.z) return;
				tMax.z += tDelta.z;
			}
			else	{
				idCurrentVoxel.y += step.y;
				if(idCurrentVoxel.y == idBound.y) return;
				tMax.y += tDelta.y;
			}

			iter++;
		}
	}
}

extern "C" void allocCUDA(HashDataStruct& hashData, const HashParams& hashParams, const DepthCameraData& depthCameraData, const DepthCameraParams& depthCameraParams, const unsigned int* d_bitMask) 
{
	const dim3 gridSize((depthCameraParams.m_imageWidth + T_PER_BLOCK - 1)/T_PER_BLOCK, (depthCameraParams.m_imageHeight + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	allocKernel<<<gridSize, blockSize>>>(hashData, depthCameraData, d_bitMask);

	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}



__global__ void fillDecisionArrayKernel(HashDataStruct hashData) 
{
	const HashParams& hashParams = c_hashParams;
	const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < hashParams.m_hashNumBuckets * HASH_BUCKET_SIZE) {
		hashData.d_hashDecision[idx] = 0;
		if (hashData.d_hash[idx].ptr != FREE_ENTRY) {
			if (hashData.isSDFBlockInCameraFrustumApprox(hashData.d_hash[idx].pos)) 
			{
				hashData.d_hashDecision[idx] = 1;	//yes
			}
		}
	}
}

extern "C" void fillDecisionArrayCUDA(HashDataStruct& hashData, const HashParams& hashParams)
{
	const dim3 gridSize((HASH_BUCKET_SIZE * hashParams.m_hashNumBuckets + (T_PER_BLOCK*T_PER_BLOCK) - 1)/(T_PER_BLOCK*T_PER_BLOCK), 1);
	const dim3 blockSize((T_PER_BLOCK*T_PER_BLOCK), 1);

	fillDecisionArrayKernel<<<gridSize, blockSize>>>(hashData);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

}

__global__ void compactifyHashKernel(HashDataStruct hashData) 
{
	const HashParams& hashParams = c_hashParams;
	const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < hashParams.m_hashNumBuckets * HASH_BUCKET_SIZE) {
		if (hashData.d_hashDecision[idx] == 1) {
			hashData.d_hashCompactified[hashData.d_hashDecisionPrefix[idx]-1] = hashData.d_hash[idx];
		}
	}
}

extern "C" void compactifyHashCUDA(HashDataStruct& hashData, const HashParams& hashParams) 
{
	const dim3 gridSize((HASH_BUCKET_SIZE * hashParams.m_hashNumBuckets + (T_PER_BLOCK*T_PER_BLOCK) - 1)/(T_PER_BLOCK*T_PER_BLOCK), 1);
	const dim3 blockSize((T_PER_BLOCK*T_PER_BLOCK), 1);

	compactifyHashKernel<<<gridSize, blockSize>>>(hashData);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

#define COMPACTIFY_HASH_THREADS_PER_BLOCK 256
//#define COMPACTIFY_HASH_SIMPLE
__global__ void compactifyHashAllInOneKernel(HashDataStruct hashData)
{
	const HashParams& hashParams = c_hashParams;
	const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
#ifdef COMPACTIFY_HASH_SIMPLE
	if (idx < hashParams.m_hashNumBuckets * HASH_BUCKET_SIZE) {
		if (hashData.d_hash[idx].ptr != FREE_ENTRY) {
			if (hashData.isSDFBlockInCameraFrustumApprox(hashData.d_hash[idx].pos))
			{
				int addr = atomicAdd(hashData.d_hashCompactifiedCounter, 1);
				hashData.d_hashCompactified[addr] = hashData.d_hash[idx];
			}
		}
	}
#else	
	__shared__ int localCounter;
	if (threadIdx.x == 0) localCounter = 0;
	__syncthreads();
	
	int addrLocal = -1;
	if (idx < hashParams.m_hashNumBuckets * HASH_BUCKET_SIZE) {
		if (hashData.d_hash[idx].ptr != FREE_ENTRY) {
			if (hashData.isSDFBlockInCameraFrustumApprox(hashData.d_hash[idx].pos))
			{
				addrLocal = atomicAdd(&localCounter, 1);
			}
		}
	}

	__syncthreads();

	__shared__ int addrGlobal;
	if (threadIdx.x == 0 && localCounter > 0) {
		addrGlobal = atomicAdd(hashData.d_hashCompactifiedCounter, localCounter);
	}
	__syncthreads();

	if (addrLocal != -1) {
		const unsigned int addr = addrGlobal + addrLocal;
		hashData.d_hashCompactified[addr] = hashData.d_hash[idx];
	}
#endif
}

extern "C" unsigned int compactifyHashAllInOneCUDA(HashDataStruct& hashData, const HashParams& hashParams)
{
	const unsigned int threadsPerBlock = COMPACTIFY_HASH_THREADS_PER_BLOCK;
	const dim3 gridSize((HASH_BUCKET_SIZE * hashParams.m_hashNumBuckets + threadsPerBlock - 1) / threadsPerBlock, 1);
	const dim3 blockSize(threadsPerBlock, 1);

	cutilSafeCall(hipMemset(hashData.d_hashCompactifiedCounter, 0, sizeof(int)));
	compactifyHashAllInOneKernel << <gridSize, blockSize >> >(hashData);
	unsigned int res = 0;
	cutilSafeCall(hipMemcpy(&res, hashData.d_hashCompactifiedCounter, sizeof(unsigned int), hipMemcpyDeviceToHost));

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	return res;
}


inline __device__ float4 bilinearFilterColor(const float2& screenPos) {
	const DepthCameraParams& cameraParams = c_depthCameraParams;
	const int imageWidth = cameraParams.m_imageWidth;
	const int imageHeight = cameraParams.m_imageHeight;
	const int2 p00 = make_int2(screenPos.x+0.5f, screenPos.y+0.5f);
	const int2 dir = sign(make_float2(screenPos.x - p00.x, screenPos.y - p00.y));

	const int2 p01 = p00 + make_int2(0.0f, dir.y);
	const int2 p10 = p00 + make_int2(dir.x, 0.0f);
	const int2 p11 = p00 + make_int2(dir.x, dir.y);

	const float alpha = (screenPos.x - p00.x)*dir.x;
	const float beta  = (screenPos.y - p00.y)*dir.y;

	float4 s0 = make_float4(0.0f, 0.0f, 0.0f, 0.0f); float w0 = 0.0f;
	if (p00.x >= 0 && p00.x < imageWidth && p00.y >= 0 && p00.y < imageHeight) { uchar4 v00_uc = tex2D(colorTextureRef, p00.x, p00.y); float4 v00 = make_float4(v00_uc.x, v00_uc.y, v00_uc.z, v00_uc.w);	if (v00.x != MINF) { s0 += (1.0f - alpha)*v00; w0 += (1.0f - alpha); } }
	if (p10.x >= 0 && p10.x < imageWidth && p10.y >= 0 && p10.y < imageHeight) { uchar4 v10_uc = tex2D(colorTextureRef, p10.x, p10.y); float4 v10 = make_float4(v10_uc.x, v10_uc.y, v10_uc.z, v10_uc.w);    if (v10.x != MINF) { s0 += alpha *v10; w0 += alpha; } }

	float4 s1 = make_float4(0.0f, 0.0f, 0.0f, 0.0f); float w1 = 0.0f;
	if (p01.x >= 0 && p01.x < imageWidth && p01.y >= 0 && p01.y < imageHeight) { uchar4 v01_uc = tex2D(colorTextureRef, p01.x, p01.y); float4 v01 = make_float4(v01_uc.x, v01_uc.y, v01_uc.z, v01_uc.w);    if (v01.x != MINF) { s1 += (1.0f - alpha)*v01; w1 += (1.0f - alpha); } }
	if (p11.x >= 0 && p11.x < imageWidth && p11.y >= 0 && p11.y < imageHeight) { uchar4 v11_uc = tex2D(colorTextureRef, p11.x, p11.y); float4 v11 = make_float4(v11_uc.x, v11_uc.y, v11_uc.z, v11_uc.w);    if (v11.x != MINF) { s1 += alpha *v11; w1 += alpha; } }

	const float4 p0 = s0/w0;
	const float4 p1 = s1/w1;

	float4 ss = make_float4(0.0f, 0.0f, 0.0f, 0.0f); float ww = 0.0f;
	if(w0 > 0.0f) { ss += (1.0f-beta)*p0; ww += (1.0f-beta); }
	if(w1 > 0.0f) { ss +=		beta *p1; ww +=		  beta ; }

	if(ww > 0.0f) return ss/ww;
	else		  return make_float4(MINF, MINF, MINF, MINF);
}

template<bool deIntegrate = false>
__global__ void integrateDepthMapKernel(HashDataStruct hashData, DepthCameraData cameraData) {
	const HashParams& hashParams = c_hashParams;
	const DepthCameraParams& cameraParams = c_depthCameraParams;

	const HashEntry& entry = hashData.d_hashCompactified[blockIdx.x];

	int3 pi_base = hashData.SDFBlockToVirtualVoxelPos(entry.pos);

	uint i = threadIdx.x;	//inside of an SDF block
	int3 pi = pi_base + make_int3(hashData.delinearizeVoxelIndex(i));
	float3 pf = hashData.virtualVoxelPosToWorld(pi);

	pf = hashParams.m_rigidTransformInverse * pf;
	uint2 screenPos = make_uint2(cameraData.cameraToKinectScreenInt(pf));


	if (screenPos.x < cameraParams.m_imageWidth && screenPos.y < cameraParams.m_imageHeight) {	//on screen

		//float depth = g_InputDepth[screenPos];
		float depth = tex2D(depthTextureRef, screenPos.x, screenPos.y);
		float4 color  = make_float4(MINF, MINF, MINF, MINF);
		if (cameraData.d_colorData) {
			uchar4 color_uc = tex2D(colorTextureRef, screenPos.x, screenPos.y);
			color = make_float4(color_uc.x, color_uc.y, color_uc.z, color_uc.w);
			//color = bilinearFilterColor(cameraData.cameraToKinectScreenFloat(pf));
		}

		if (color.x != MINF && depth != MINF) { // valid depth and color
		//if (depth != MINF) {	//valid depth

			if (depth < hashParams.m_maxIntegrationDistance) {
				float depthZeroOne = cameraData.cameraToKinectProjZ(depth);

				float sdf = depth - pf.z;
				float truncation = hashData.getTruncation(depth);
				//if (sdf > -truncation) 
				if (abs(sdf) < truncation)
				{
					if (sdf >= 0.0f) {
						sdf = fminf(truncation, sdf);
					} else {
						sdf = fmaxf(-truncation, sdf);
					}

					float weightUpdate = max(hashParams.m_integrationWeightSample * 1.5f * (1.0f-depthZeroOne), 1.0f);
					weightUpdate = 1.0f;	//TODO remove that again

					Voxel curr;	//construct current voxel
					curr.sdf = sdf;
					curr.weight = weightUpdate;

					if (cameraData.d_colorData) {
						curr.color = make_uchar4(color.x, color.y, color.z, 255);
					} else {
						curr.color = make_uchar4(0,255,0,0);
					}

					uint idx = entry.ptr + i;
					
					const Voxel& oldVoxel = hashData.d_SDFBlocks[idx];
					Voxel newVoxel;

					float3 oldColor = make_float3(oldVoxel.color.x, oldVoxel.color.y, oldVoxel.color.z);
					float3 currColor = make_float3(curr.color.x, curr.color.y, curr.color.z);

					if (!deIntegrate) {	//integration
						//hashData.combineVoxel(hashData.d_SDFBlocks[idx], curr, newVoxel);
						float3 res;
						if (oldVoxel.weight == 0) res = currColor;
						//else res = (currColor + oldColor) / 2;
						else res = 0.2f * currColor + 0.8f * oldColor;
						//float3 res = (currColor*curr.weight + oldColor*oldVoxel.weight) / (curr.weight + oldVoxel.weight);
						res = make_float3(round(res.x), round(res.y), round(res.z));
						res = fmaxf(make_float3(0.0f), fminf(res, make_float3(254.5f)));
						//newVoxel.color.x = (uchar)(res.x + 0.5f);	newVoxel.color.y = (uchar)(res.y + 0.5f);	newVoxel.color.z = (uchar)(res.z + 0.5f);
						newVoxel.color = make_uchar4(res.x, res.y, res.z, 255);
						newVoxel.sdf = (curr.sdf*curr.weight + oldVoxel.sdf*oldVoxel.weight) / (curr.weight + oldVoxel.weight);
						newVoxel.weight = min((float)c_hashParams.m_integrationWeightMax, curr.weight + oldVoxel.weight);
					}
					else {				//deintegration
						//float3 res = 2 * c0 - c1;
						float3 res = (oldColor*oldVoxel.weight - currColor*curr.weight) / (oldVoxel.weight - curr.weight);
						res = make_float3(round(res.x), round(res.y), round(res.z));
						res = fmaxf(make_float3(0.0f), fminf(res, make_float3(254.5f)));
						//newVoxel.color.x = (uchar)(res.x + 0.5f);	newVoxel.color.y = (uchar)(res.y + 0.5f);	newVoxel.color.z = (uchar)(res.z + 0.5f);
						newVoxel.color = make_uchar4(res.x, res.y, res.z, 255);
						newVoxel.sdf = (oldVoxel.sdf*oldVoxel.weight - curr.sdf*curr.weight) / (oldVoxel.weight - curr.weight);
						newVoxel.weight = max(0.0f, oldVoxel.weight - curr.weight);
						if (newVoxel.weight <= 0.001f) {
							newVoxel.sdf = 0.0f;
							newVoxel.color = make_uchar4(0,0,0,0);
							newVoxel.weight = 0.0f;
						}
					}

					hashData.d_SDFBlocks[idx] = newVoxel;
				}
			}
		}
	}
}


extern "C" void integrateDepthMapCUDA(HashDataStruct& hashData, const HashParams& hashParams, const DepthCameraData& depthCameraData, const DepthCameraParams& depthCameraParams)
{
	const unsigned int threadsPerBlock = SDF_BLOCK_SIZE*SDF_BLOCK_SIZE*SDF_BLOCK_SIZE;
	const dim3 gridSize(hashParams.m_numOccupiedBlocks, 1);
	const dim3 blockSize(threadsPerBlock, 1);

	integrateDepthMapKernel<false> <<<gridSize, blockSize>>>(hashData, depthCameraData);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

extern "C" void deIntegrateDepthMapCUDA(HashDataStruct& hashData, const HashParams& hashParams, const DepthCameraData& depthCameraData, const DepthCameraParams& depthCameraParams)
{
	const unsigned int threadsPerBlock = SDF_BLOCK_SIZE*SDF_BLOCK_SIZE*SDF_BLOCK_SIZE;
	const dim3 gridSize(hashParams.m_numOccupiedBlocks, 1);
	const dim3 blockSize(threadsPerBlock, 1);

	integrateDepthMapKernel<true> <<<gridSize, blockSize >>>(hashData, depthCameraData);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}



__global__ void starveVoxelsKernel(HashDataStruct hashData) {

	const uint idx = blockIdx.x;
	const HashEntry& entry = hashData.d_hashCompactified[idx];

	//is typically exectued only every n'th frame
	int weight = hashData.d_SDFBlocks[entry.ptr + threadIdx.x].weight;
	weight = max(0, weight-1);	
	hashData.d_SDFBlocks[entry.ptr + threadIdx.x].weight = weight;
}

extern "C" void starveVoxelsKernelCUDA(HashDataStruct& hashData, const HashParams& hashParams)
{
	const unsigned int threadsPerBlock = SDF_BLOCK_SIZE*SDF_BLOCK_SIZE*SDF_BLOCK_SIZE;
	const dim3 gridSize(hashParams.m_numOccupiedBlocks, 1);
	const dim3 blockSize(threadsPerBlock, 1);

	starveVoxelsKernel<<<gridSize, blockSize>>>(hashData);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}


//__shared__ float	shared_MinSDF[SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE / 2];
__shared__ uint		shared_MaxWeight[SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE / 2];


__global__ void garbageCollectIdentifyKernel(HashDataStruct hashData) {

	const unsigned int hashIdx = blockIdx.x;
	const HashEntry& entry = hashData.d_hashCompactified[hashIdx];
	
	//uint h = hashData.computeHashPos(entry.pos);
	//hashData.d_hashDecision[hashIdx] = 1;
	//if (hashData.d_hashBucketMutex[h] == LOCK_ENTRY)	return;

	//if (entry.ptr == FREE_ENTRY) return; //should never happen since we did compactify before
	//const uint linBlockSize = SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	const unsigned int idx0 = entry.ptr + 2*threadIdx.x+0;
	const unsigned int idx1 = entry.ptr + 2*threadIdx.x+1;

	Voxel v0 = hashData.d_SDFBlocks[idx0];
	Voxel v1 = hashData.d_SDFBlocks[idx1];

	//if (v0.weight == 0)	v0.sdf = PINF;
	//if (v1.weight == 0)	v1.sdf = PINF;

	//shared_MinSDF[threadIdx.x] = min(fabsf(v0.sdf), fabsf(v1.sdf));	//init shared memory
	shared_MaxWeight[threadIdx.x] = max(v0.weight, v1.weight);
		
#pragma unroll 1
	for (uint stride = 2; stride <= blockDim.x; stride <<= 1) {
		__syncthreads();
		if ((threadIdx.x  & (stride-1)) == (stride-1)) {
			//shared_MinSDF[threadIdx.x] = min(shared_MinSDF[threadIdx.x-stride/2], shared_MinSDF[threadIdx.x]);
			shared_MaxWeight[threadIdx.x] = max(shared_MaxWeight[threadIdx.x-stride/2], shared_MaxWeight[threadIdx.x]);
		}
	}

	__syncthreads();

	if (threadIdx.x == blockDim.x - 1) {
		//float minSDF = shared_MinSDF[threadIdx.x];
		uint maxWeight = shared_MaxWeight[threadIdx.x];

		//float t = hashData.getTruncation(c_depthCameraParams.m_sensorDepthWorldMax);
		//if (minSDF >= t || maxWeight == 0) {
		if (maxWeight == 0) {
			hashData.d_hashDecision[hashIdx] = 1;
		} else {
			hashData.d_hashDecision[hashIdx] = 0; 
		}
	}
}
 
extern "C" void garbageCollectIdentifyCUDA(HashDataStruct& hashData, const HashParams& hashParams) {
	
	const unsigned int threadsPerBlock = SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE / 2;
	const dim3 gridSize(hashParams.m_numOccupiedBlocks, 1);
	const dim3 blockSize(threadsPerBlock, 1);

	garbageCollectIdentifyKernel<<<gridSize, blockSize>>>(hashData);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}


__global__ void garbageCollectFreeKernel(HashDataStruct hashData) {

	//const uint hashIdx = blockIdx.x;
	const uint hashIdx = blockIdx.x*blockDim.x + threadIdx.x;


	if (hashIdx < c_hashParams.m_numOccupiedBlocks && hashData.d_hashDecision[hashIdx] != 0) {	//decision to delete the hash entry

		const HashEntry& entry = hashData.d_hashCompactified[hashIdx];
		//if (entry.ptr == FREE_ENTRY) return; //should never happen since we did compactify before

		if (hashData.deleteHashEntryElement(entry.pos)) {	//delete hash entry from hash (and performs heap append)
			const uint linBlockSize = SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

			#pragma unroll 1
			for (uint i = 0; i < linBlockSize; i++) {	//clear sdf block: CHECK TODO another kernel?
				hashData.deleteVoxel(entry.ptr + i);
			}
		}
	}
}


extern "C" void garbageCollectFreeCUDA(HashDataStruct& hashData, const HashParams& hashParams) {
	
	const unsigned int threadsPerBlock = T_PER_BLOCK*T_PER_BLOCK;
	const dim3 gridSize((hashParams.m_numOccupiedBlocks + threadsPerBlock - 1) / threadsPerBlock, 1);
	const dim3 blockSize(threadsPerBlock, 1);

	garbageCollectFreeKernel<<<gridSize, blockSize>>>(hashData);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

