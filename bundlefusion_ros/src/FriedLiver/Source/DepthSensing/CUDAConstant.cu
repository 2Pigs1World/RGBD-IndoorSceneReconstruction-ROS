
#include "CUDAHashParams.h"
#include "CUDARayCastParams.h"
#include "CUDADepthCameraParams.h"

__constant__ HashParams c_hashParams;
__constant__ RayCastParams c_rayCastParams;
__constant__ DepthCameraParams c_depthCameraParams;

extern "C" void updateConstantHashParams(const HashParams& params) {

	size_t size;
	cutilSafeCall(hipGetSymbolSize(&size, HIP_SYMBOL(c_hashParams)));
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_hashParams), &params, size, 0, hipMemcpyHostToDevice));
	
#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	}


extern "C" void updateConstantRayCastParams(const RayCastParams& params) {
	
	size_t size;
	cutilSafeCall(hipGetSymbolSize(&size, HIP_SYMBOL(c_rayCastParams)));
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_rayCastParams), &params, size, 0, hipMemcpyHostToDevice));
	
#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

}

extern "C" void updateConstantDepthCameraParams(const DepthCameraParams& params) {
	
	size_t size;
	cutilSafeCall(hipGetSymbolSize(&size, HIP_SYMBOL(c_depthCameraParams)));
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_depthCameraParams), &params, size, 0, hipMemcpyHostToDevice));
	
#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

}

